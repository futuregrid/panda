#include "hip/hip_runtime.h"
/*	
	Copyright 2012 The Trustees of Indiana University.  All rights reserved.
	CGL MapReduce Framework on GPUs and CPUs
	
	Code Name: Panda 
	
	File: map.cu 
	First Version:		2012-07-01 V0.1
	Current Version:	2012-09-01 V0.3	
	Last Updates:		2012-09-02

	Developer: Hui Li (lihui@indiana.edu)

	This is the source code for Panda, a MapReduce runtime on GPUs and CPUs.

 */

#ifndef __USER_CU__
#define __USER_CU__

#include "UserAPI.h"
#include "Panda.h"


void cpu_map(void *KEY, void*VAL, int keySize, int valSize, cpu_context *d_g_state, int map_task_idx){
		
		int wsize = 0;
		char *start;
		char *p = (char *)VAL;

		while(1)
		{
			start = p;
			for(;*p>='A' && *p<='Z';p++);
			*p='\0';
			++p;
			wsize=(int)(p-start);
			if (wsize>6){
				char *wkey = (char *) malloc (wsize);
				memcpy(wkey,start,wsize);
				
				int *wc = (int *) malloc (sizeof(int));
				*wc=1;
				
				CPUEmitMapOutput(wkey, wc, wsize, sizeof(int), d_g_state, map_task_idx);
			}//if
			valSize = valSize - wsize;
			if(valSize<=0)
				break;
		}//while
}//map2

__device__ void gpu_core_map(void *KEY, void*VAL, int keySize, int valSize, panda_gpu_context *pgc, int map_task_idx){

		
		int wsize = 0;
		char *start;
		char *p = (char *)VAL;
		int *wc = (int *) malloc (sizeof(int));
		*wc = 1;

		while(1)
		{
			start = p;
			for(; *p>='A' && *p<='Z'; p++);

			*p='\0';
			++p;
			wsize=(int)(p-start);
			if (wsize>6){
				char *wkey = start;
				GPUEmitMapOutput(wkey, wc, wsize, sizeof(int), pgc, map_task_idx);
			}//if
			valSize = valSize - wsize;
			if(valSize<=0)
				break;
		}//while

		printf("gpu_core\n");
		
		__syncthreads();
		
}//map2

__device__ void gpu_core_map(void *KEY, void*VAL, int keySize, int valSize, gpu_context *d_g_state, int map_task_idx){

		int wsize = 0;
		char *start;
		char *p = (char *)VAL;
		int *wc = (int *) malloc (sizeof(int));
		*wc = 1;

		while(1)
		{
			start = p;
			for(; *p>='A' && *p<='Z'; p++);

			*p='\0';
			++p;
			wsize=(int)(p-start);
			if (wsize>6){
				char *wkey = start;
				GPUEmitMapOutput(wkey, wc, wsize, sizeof(int), d_g_state, map_task_idx);
			}//if
			valSize = valSize - wsize;
			if(valSize<=0)
				break;
		}//while
		
		__syncthreads();
		
}//map2

void cpu_combiner(void *KEY, val_t* VAL, int keySize, int valCount, cpu_context *d_g_state, int map_task_idx){
		
		//int *count = (int *) malloc (sizeof(int));
		int count = 0;
		for (int i=0;i<valCount;i++){
			 count += *((int *)(VAL[i].val));
		}//for

		CPUEmitCombinerOutput(KEY,&count,keySize,sizeof(int),d_g_state, map_task_idx);
		
}//reduce2

__device__ void gpu_combiner(void *KEY, val_t* VAL, int keySize, int valCount, panda_gpu_context *pgc, int map_task_idx){
		
		//int *count = (int *) malloc (sizeof(int));

		int count = 0;
		for (int i=0;i<valCount;i++){
			 count += *((int *)(VAL[i].val));
		}//

		GPUEmitCombinerOutput(KEY,&count,keySize,sizeof(int),pgc, map_task_idx);
		
}//reduce2



__device__ void gpu_combiner(void *KEY, val_t* VAL, int keySize, int valCount, gpu_context *d_g_state, int map_task_idx){
		
		//int *count = (int *) malloc (sizeof(int));

		int count = 0;
		for (int i=0;i<valCount;i++){
			 count += *((int *)(VAL[i].val));
		}//

		GPUEmitCombinerOutput(KEY,&count,keySize,sizeof(int),d_g_state, map_task_idx);
		
}//reduce2


__device__ void panda_gpu_reduce(void *KEY, val_t* VAL, int keySize, int valCount, panda_gpu_context pgc){

		int count = 0;
		for (int i=0;i<valCount;i++){
			count += *(int *)(VAL[i].val);
		}//
		
		//GPUEmitReduceOuput(KEY,&count,keySize,sizeof(int),&pgc);
		PandaGPUEmitReduceOutput(KEY,&count,keySize,sizeof(int),&pgc);
		
}//reduce2

__device__ void gpu_reduce(void *KEY, val_t* VAL, int keySize, int valCount, gpu_context d_g_state){

		int count = 0;
		for (int i=0;i<valCount;i++){
			count += *(int *)(VAL[i].val);
		}//
		
		GPUEmitReduceOuput(KEY,&count,keySize,sizeof(int),&d_g_state);
		
}//reduce2

void cpu_reduce(void *KEY, val_t* VAL, int keySize, int valCount, cpu_context* d_g_state){

		int count = 0;
		for (int i=0;i<valCount;i++){
			count += *(int *)(VAL[i].val);
		}//
		
		CPUEmitReduceOutput(KEY,&count,keySize,sizeof(int),d_g_state);
		
}//reduce2



int cpu_compare(const void *d_a, int len_a, const void *d_b, int len_b)
{
	char* word1 = (char*)d_a;
	char* word2 = (char*)d_b;

	for (; *word1 != '\0' && *word2 != '\0' && *word1 == *word2; word1++, word2++);
	if (*word1 > *word2) return 1;
	if (*word1 < *word2) return -1;

	return 0;
}

__device__ int gpu_compare(const void *d_a, int len_a, const void *d_b, int len_b)
{
	char* word1 = (char*)d_a;
	char* word2 = (char*)d_b;

	for (; *word1 != '\0' && *word2 != '\0' && *word1 == *word2; word1++, word2++);
	if (*word1 > *word2) return 1;
	if (*word1 < *word2) return -1;

	return 0;
}


#endif //__MAP_CU__