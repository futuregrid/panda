#include "hip/hip_runtime.h"
/*
	
	Copyright 2012 The Trustees of Indiana University.  All rights reserved.
	Panda: co-processing SPMD computations on GPUs and CPUs.
	
	File: PandaLib.cu
	First Version:		2012-07-01 V0.1
	Last UPdates: 		2018-04-28 v0.61	
	Developer: Hui Li (huili@ruijie.com.cn)

 */

#ifndef __PANDALIB_CU__
#define __PANDALIB_CU__

#include "Panda.h"
#include "Global.h"
#include "PandaAPI.h"

extern int gCommRank;

__global__ void ExecutePandaGPUMapPartitioner(panda_gpu_context pgc)
{
	//ShowLog2("gridDim.x:%d gridDim.y:%d gridDim.z:%d blockDim.x:%d blockDim.y:%d blockDim.z:%d blockIdx.x:%d blockIdx.y:%d blockIdx.z:%d\n",
	// 		gridDim.x,gridDim.y,gridDim.z,blockDim.x,blockDim.y,blockDim.z,blockIdx.x,blockIdx.y,blockIdx.z);
	int num_records_per_thread = (pgc.input_key_vals.num_input_record + (gridDim.x*blockDim.x*blockDim.y)-1)/(gridDim.x*blockDim.x*blockDim.y);
	int block_start_idx = num_records_per_thread * blockIdx.x * blockDim.x * blockDim.y;
	int thread_start_idx = block_start_idx 
		+ ((threadIdx.y*blockDim.x + threadIdx.x)/STRIDE)*num_records_per_thread*STRIDE
		+ ((threadIdx.y*blockDim.x + threadIdx.x)%STRIDE);

	int thread_end_idx = thread_start_idx + num_records_per_thread*STRIDE;
	if (thread_end_idx > pgc.input_key_vals.num_input_record)
		thread_end_idx = pgc.input_key_vals.num_input_record;

	if (thread_start_idx >= thread_end_idx)
		return;

	//if(TID==0) ShowWarn("hi 0 -- num_records_per_thread:%d",num_records_per_thread);

	int buddy_arr_len = num_records_per_thread;
	int * int_arr = (int*)malloc((4+buddy_arr_len)*sizeof(int));
	if(int_arr==NULL){ GpuShowError("there is not enough GPU memory\n"); return;}

	int *shared_arr_len = int_arr;
	int *shared_buff_len = int_arr+1;
	int *shared_buff_pos = int_arr+2;
	//int *num_buddy = int_arr+3;
	int *buddy = int_arr+4;
	(*shared_buff_len) = SHARED_BUFF_LEN;
	(*shared_arr_len) = 0;
	(*shared_buff_pos) = 0;

	char * buff = (char *)malloc(sizeof(char)*(*shared_buff_len));
	keyval_arr_t *kv_arr_t_arr = (keyval_arr_t *)malloc(sizeof(keyval_arr_t)*(thread_end_idx-thread_start_idx+STRIDE-1)/STRIDE);
	int index = 0;
	
	for(int idx = thread_start_idx; idx < thread_end_idx; idx += STRIDE){
			buddy[index] = idx;
			index ++;
	}//for
	index = 0;
	for(int map_task_idx = thread_start_idx; map_task_idx < thread_end_idx; map_task_idx += STRIDE){

		keyval_arr_t *kv_arr_t = (keyval_arr_t *)&(kv_arr_t_arr[index]);
		index++;
		kv_arr_t->shared_buff = buff;
		kv_arr_t->shared_arr_len = shared_arr_len;
		kv_arr_t->shared_buff_len = shared_buff_len;
		kv_arr_t->shared_buff_pos = shared_buff_pos;
		kv_arr_t->shared_buddy = buddy;
		kv_arr_t->shared_buddy_len = buddy_arr_len;
		kv_arr_t->arr = NULL;
		kv_arr_t->arr_len = 0;
		
		pgc.intermediate_key_vals.d_intermediate_keyval_arr_arr_p[map_task_idx] = kv_arr_t;

	}//for
}

void StartPandaGPUMapPartitioner(panda_gpu_context pgc, dim3 grids, dim3 blocks)
{
   	ExecutePandaGPUMapPartitioner<<<grids,blocks>>>(pgc);
}

void* ExecutePandaCPUMapThread(void * ptr)
{

	panda_cpu_task_info_t *panda_cpu_task_info = (panda_cpu_task_info_t *)ptr;
	panda_cpu_context  *pcc = (panda_cpu_context *) (panda_cpu_task_info->pcc);
	//panda_node_context *pnc = (panda_node_context *)(panda_cpu_task_info->pnc);
	
	int start_row_idx	=	panda_cpu_task_info->start_row_idx;
	int end_row_idx		=	panda_cpu_task_info->end_row_idx;

	if(end_row_idx<=start_row_idx) 	return NULL;
	
	char *buff		=	(char *)malloc(sizeof(char)*CPU_SHARED_BUFF_SIZE);
	int *int_arr	=	(int *)malloc(sizeof(int)*(end_row_idx - start_row_idx + 3));
	int *buddy		=	int_arr+3;
	
	int buddy_len	=	end_row_idx	- start_row_idx;
	for (int i=0;i<buddy_len;i++){
		buddy [i]	=	i + start_row_idx;
	}//for
	
	for (int map_idx = start_row_idx; map_idx < end_row_idx; map_idx++){

		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buff)		= buff;
		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buff_len) = int_arr;
		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buff_pos) = int_arr+1;
		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_arr_len)	= int_arr+2;
		
		*(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buff_len)	= CPU_SHARED_BUFF_SIZE;
		*(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buff_pos)	= 0;
		*(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_arr_len)		= 0;
		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buddy)		= buddy;
		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buddy_len)	= buddy_len;

	}//for

	for (int map_idx = panda_cpu_task_info->start_row_idx; map_idx < panda_cpu_task_info->end_row_idx; map_idx++){
		keyval_t *kv_p = (keyval_t *)(&(pcc->input_key_vals.input_keyval_arr[map_idx]));
		panda_cpu_map(kv_p->key,kv_p->val,kv_p->keySize,kv_p->valSize,pcc,map_idx);
	}//for
	
	//ShowLog("CPU_GROUP_ID:[%d] Done :%d tasks",d_g_state->cpu_group_id, panda_cpu_task_info->end_row_idx - panda_cpu_task_info->start_row_idx);
	return NULL;
}//int 


void ExecutePandaReduceTasksOnGPU(panda_gpu_context *pgc)
{
	if (pgc->sorted_key_vals.d_sorted_keyvals_arr_len <= 0)
	return;

	hipDeviceSynchronize(); 
	pgc->reduced_key_vals.d_reduced_keyval_arr_len = pgc->sorted_key_vals.d_sorted_keyvals_arr_len;

	hipMalloc((void **)&(pgc->reduced_key_vals.d_reduced_keyval_arr), 
		sizeof(keyval_t)*pgc->reduced_key_vals.d_reduced_keyval_arr_len);

	pgc->output_key_vals.totalKeySize = 0;
	pgc->output_key_vals.totalValSize = 0;
	pgc->output_key_vals.h_reduced_keyval_arr_len = pgc->reduced_key_vals.d_reduced_keyval_arr_len;
	pgc->output_key_vals.h_reduced_keyval_arr = (keyval_t*)(malloc(sizeof(keyval_t)*pgc->output_key_vals.h_reduced_keyval_arr_len));
	
	hipDeviceSynchronize(); 
	int numGPUCores = getGPUCoresNum();
	dim3 blocks(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
	int numBlocks = (numGPUCores*16+(blocks.x*blocks.y)-1)/(blocks.x*blocks.y);
        dim3 grids(numBlocks, 1);
	//int total_gpu_threads = (grids.x*grids.y*blocks.x*blocks.y);
	pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_len = pgc->reduced_key_vals.d_reduced_keyval_arr_len;
	
	ShowLog("[ExecutePandaReduceTasksOnGPU] reduce len:%d intermediate len:%d output len:%d sorted keySize%d: sorted valSize:%d",
		pgc->reduced_key_vals.d_reduced_keyval_arr_len, 
		pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_len,
		pgc->output_key_vals.h_reduced_keyval_arr_len,
		pgc->sorted_key_vals.totalKeySize, 
		pgc->sorted_key_vals.totalValSize);

	PandaReducePartitioner<<<grids,blocks>>>(*pgc);

	hipMemcpy(pgc->output_key_vals.h_reduced_keyval_arr,
		pgc->reduced_key_vals.d_reduced_keyval_arr,
		sizeof(keyval_t)*pgc->reduced_key_vals.d_reduced_keyval_arr_len,
		hipMemcpyDeviceToHost);

	for (int i = 0; i<pgc->reduced_key_vals.d_reduced_keyval_arr_len; i++){
		pgc->output_key_vals.totalKeySize += (pgc->output_key_vals.h_reduced_keyval_arr[i].keySize+3)/4*4;
		pgc->output_key_vals.totalValSize += (pgc->output_key_vals.h_reduced_keyval_arr[i].valSize+3)/4*4;
	}//for
	
	//ShowLog("Output total keySize:%f KB valSize:%f KB\n",(float)(pgc->output_key_vals.totalKeySize)/1024.0,(float)(pgc->output_key_vals.totalValSize)/1024.0);

	pgc->output_key_vals.h_KeyBuff = malloc(sizeof(char)*pgc->output_key_vals.totalKeySize);
	pgc->output_key_vals.h_ValBuff = malloc(sizeof(char)*pgc->output_key_vals.totalValSize);

	hipMalloc(&(pgc->output_key_vals.d_KeyBuff), sizeof(char)*pgc->output_key_vals.totalKeySize );
	hipMalloc(&(pgc->output_key_vals.d_ValBuff), sizeof(char)*pgc->output_key_vals.totalValSize );

	ShowLog("[copyDataFromDevice2Host4Reduce] Output total keySize:%f KB valSize:%f KB\n",(float)(pgc->output_key_vals.totalKeySize)/1024.0,(float)(pgc->output_key_vals.totalValSize)/1024.0);

	copyDataFromDevice2Host4Reduce<<<grids,blocks>>>(*pgc);

	hipMemcpy(
			pgc->output_key_vals.h_KeyBuff,
			pgc->output_key_vals.d_KeyBuff,
			pgc->output_key_vals.totalKeySize,
		hipMemcpyDeviceToHost);

	hipMemcpy(
		pgc->output_key_vals.h_ValBuff,
		pgc->output_key_vals.d_ValBuff,
		pgc->output_key_vals.totalValSize,
		hipMemcpyDeviceToHost);

	int val_pos, key_pos;
	val_pos = key_pos = 0;
	void *val, *key;

	for (int i = 0; i<pgc->output_key_vals.h_reduced_keyval_arr_len; i++){
		
		val = (char *)pgc->output_key_vals.h_ValBuff + val_pos;
		key = (char *)pgc->output_key_vals.h_KeyBuff + key_pos;
		pgc->output_key_vals.h_reduced_keyval_arr[i].key = key;
		pgc->output_key_vals.h_reduced_keyval_arr[i].val = val;
		ShowLog("key:%s val:%d",(char *)key,*(int*)val);

		val_pos += (pgc->output_key_vals.h_reduced_keyval_arr[i].valSize+3)/4*4;
		key_pos += (pgc->output_key_vals.h_reduced_keyval_arr[i].keySize+3)/4*4;

	}//for

	//TODO
	hipDeviceSynchronize(); 

}//void


__global__ void PandaReducePartitioner(panda_gpu_context pgc)
{
	//ShowError("ReducePartitioner Panda_GPU_Context");
	int num_records_per_thread = (pgc.sorted_key_vals.d_sorted_keyvals_arr_len + (gridDim.x*blockDim.x*blockDim.y)-1)/(gridDim.x*blockDim.x*blockDim.y);

	int block_start_idx = num_records_per_thread * blockIdx.x * blockDim.x * blockDim.y;
	int thread_start_idx = block_start_idx 
		+ ((threadIdx.y*blockDim.x + threadIdx.x)/STRIDE)*num_records_per_thread*STRIDE
		+ ((threadIdx.y*blockDim.x + threadIdx.x)%STRIDE);

	int thread_end_idx = thread_start_idx + num_records_per_thread;//*STRIDE;
	
	if (thread_end_idx > pgc.sorted_key_vals.d_sorted_keyvals_arr_len)
		thread_end_idx = pgc.sorted_key_vals.d_sorted_keyvals_arr_len;

	if (thread_start_idx >= thread_end_idx)
		return;

	int start_idx, end_idx;
	for(int reduce_task_idx=thread_start_idx; reduce_task_idx < thread_end_idx; reduce_task_idx++/*=STRIDE*/){

		if (reduce_task_idx==0)
			start_idx = 0;
		else
			start_idx = pgc.sorted_key_vals.d_pos_arr_4_sorted_keyval_pos_arr[reduce_task_idx-1];
		end_idx = pgc.sorted_key_vals.d_pos_arr_4_sorted_keyval_pos_arr[reduce_task_idx];
		val_t *val_t_arr = (val_t*)malloc(sizeof(val_t)*(end_idx-start_idx));
		
		int keySize = pgc.sorted_key_vals.d_keyval_pos_arr[start_idx].keySize;
		int keyPos = pgc.sorted_key_vals.d_keyval_pos_arr[start_idx].keyPos;
		void *key = (char*)pgc.sorted_key_vals.d_sorted_keys_shared_buff+keyPos;
				
		for (int index = start_idx;index<end_idx;index++){
			int valSize = pgc.sorted_key_vals.d_keyval_pos_arr[index].valSize;
			int valPos = pgc.sorted_key_vals.d_keyval_pos_arr[index].valPos;
			val_t_arr[index-start_idx].valSize = valSize;
			val_t_arr[index-start_idx].val = (char*)pgc.sorted_key_vals.d_sorted_vals_shared_buff + valPos;
		}   //for
		if( end_idx - start_idx == 0) {
		GpuShowError("gpu_reduce valCount ==0");
		}//if
		else panda_gpu_reduce(key, val_t_arr, keySize, end_idx-start_idx, pgc);
	}//for
}

__global__ void PandaRunGPUMapTasks(panda_gpu_context pgc, int curIter, int totalIter)
{

	//ShowLog("gridDim.x:%d gridDim.y:%d gridDim.z:%d blockDim.x:%d blockDim.y:%d blockDim.z:%d blockIdx.x:%d blockIdx.y:%d blockIdx.z:%d\n",
	//  gridDim.x,gridDim.y,gridDim.z,blockDim.x,blockDim.y,blockDim.z,blockIdx.x,blockIdx.y,blockIdx.z);
	int num_records_per_thread = (pgc.input_key_vals.num_input_record + (gridDim.x*blockDim.x*blockDim.y)-1)/(gridDim.x*blockDim.x*blockDim.y);
	int block_start_idx = num_records_per_thread * blockIdx.x * blockDim.x * blockDim.y;
	int thread_start_idx = block_start_idx 
		+ ((threadIdx.y*blockDim.x + threadIdx.x)/STRIDE)*num_records_per_thread*STRIDE
		+ ((threadIdx.y*blockDim.x + threadIdx.x)%STRIDE);
	//ShowLog("num_records_per_thread:%d block_start_idx:%d gridDim.x:%d gridDim.y:%d gridDim.z:%d blockDim.x:%d blockDim.y:%d blockDim.z:%d",num_records_per_thread, block_start_idx, gridDim.x,gridDim.y,gridDim.z,blockDim.x,blockDim.y,blockDim.z);
	int thread_end_idx = thread_start_idx + num_records_per_thread*STRIDE;
	if (thread_end_idx > pgc.input_key_vals.num_input_record)
		thread_end_idx = pgc.input_key_vals.num_input_record;
	if (thread_start_idx + curIter*STRIDE >= thread_end_idx)
		return;
	for(int map_task_idx = thread_start_idx + curIter*STRIDE; map_task_idx < thread_end_idx; map_task_idx += totalIter*STRIDE){
		char *key = (char *)(pgc.input_key_vals.d_input_keys_shared_buff) + pgc.input_key_vals.d_input_keyval_pos_arr[map_task_idx].keyPos;
		char *val = (char *)(pgc.input_key_vals.d_input_vals_shared_buff) + pgc.input_key_vals.d_input_keyval_pos_arr[map_task_idx].valPos;
		int valSize = pgc.input_key_vals.d_input_keyval_pos_arr[map_task_idx].valSize;
		int keySize = pgc.input_key_vals.d_input_keyval_pos_arr[map_task_idx].keySize;
		/////////////////////////////////////////////////////////////////////
		panda_gpu_core_map(key, val, keySize, valSize, &pgc, map_task_idx);//
		/////////////////////////////////////////////////////////////////////
	}//for

	keyval_arr_t *kv_arr_p = pgc.intermediate_key_vals.d_intermediate_keyval_arr_arr_p[thread_start_idx];
	//char *shared_buff = (char *)(kv_arr_p->shared_buff);
	//int shared_arr_len = *kv_arr_p->shared_arr_len;
	//int shared_buff_len = *kv_arr_p->shared_buff_len;
	pgc.intermediate_key_vals.d_intermediate_keyval_total_count[thread_start_idx] = *kv_arr_p->shared_arr_len;
	//printf("CUDA Debug thread_start_idx:%d  total_count:%d\n",thread_start_idx,*kv_arr_p->shared_arr_len);
	__syncthreads();
}//GPUMapPartitioner

void *RunPandaCPUCombinerThread(void *ptr){

	panda_cpu_task_info_t *panda_cpu_task_info = (panda_cpu_task_info_t *)ptr;
	panda_cpu_context *pcc = (panda_cpu_context *)(panda_cpu_task_info->pcc); 
	bool local_combiner = false;

	int start_idx = panda_cpu_task_info->start_row_idx;
	keyval_arr_t *kv_arr_p = (keyval_arr_t *)&(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[start_idx]);
	int unmerged_shared_arr_len = *kv_arr_p->shared_arr_len;
        char *shared_buff = kv_arr_p->shared_buff;
        int shared_buff_len = *kv_arr_p->shared_buff_len;

	val_t *val_t_arr = (val_t *)malloc(sizeof(val_t)*unmerged_shared_arr_len);
	if (val_t_arr == NULL) ShowError("there is no enough memory");
	int num_keyval_pairs_after_combiner = 0;
	int total_intermediate_keyvalue_pairs = 0;

	for (int i = 0; i < unmerged_shared_arr_len; i++){

		keyval_pos_t *head_kv_p = (keyval_pos_t *)(shared_buff + shared_buff_len - sizeof(keyval_pos_t)*(unmerged_shared_arr_len-i));
		keyval_pos_t *first_kv_p = head_kv_p;

		if (first_kv_p->next_idx != _MAP)
			continue;

		int iKeySize = first_kv_p->keySize;
		char *iKey = shared_buff + first_kv_p->keyPos;
		//char *iVal = shared_buff + first_kv_p->valPos;
		if((first_kv_p->keyPos%4!=0)||(first_kv_p->valPos%4!=0)){
			ShowError("keyPos or valPos is not aligned with 4 bytes, results could be wrong");
		}
	
		int index = 0;
		(val_t_arr[index]).valSize = first_kv_p->valSize;
		(val_t_arr[index]).val = (char*)shared_buff + first_kv_p->valPos;

		ShowLog("key:%s val:%d",iKey,*(int *)(val_t_arr[index]).val);
		for (int j=i+1;j<unmerged_shared_arr_len;j++){

			keyval_pos_t *next_kv_p = (keyval_pos_t *)((char *)shared_buff + shared_buff_len - sizeof(keyval_pos_t)*(unmerged_shared_arr_len-j));
			char *jKey = (char *)shared_buff+next_kv_p->keyPos;
			int jKeySize = next_kv_p->keySize;
		
			if (!local_combiner||panda_cpu_compare(iKey,iKeySize,jKey,jKeySize)!=0){
				continue;
			}
			index++;
			first_kv_p->next_idx = j;
			first_kv_p = next_kv_p;
			(val_t_arr[index]).valSize = next_kv_p->valSize;
			(val_t_arr[index]).val = (char*)shared_buff + next_kv_p->valPos;
		}

		int valCount = index+1;
		total_intermediate_keyvalue_pairs += valCount;
		if(valCount>1){
			panda_cpu_combiner(iKey,val_t_arr,iKeySize,(valCount),pcc,start_idx);
		}//int
		else{
			first_kv_p->next_idx = _COMBINE;
			first_kv_p->task_idx = start_idx;
		}
		num_keyval_pairs_after_combiner++;
	}//for
	free(val_t_arr);
	pcc->intermediate_key_vals.intermediate_keyval_total_count[start_idx] = num_keyval_pairs_after_combiner;
	/*
	ShowLog("CPU_GROUP_ID:[%d] Map_Idx:%d  Done:%d Combiner: %d => %d Compress Ratio:%f",
		d_g_state->cpu_group_id, 
		panda_cpu_task_info->start_row_idx,
		panda_cpu_task_info->end_row_idx - panda_cpu_task_info->start_row_idx, 
		total_intermediate_keyvalue_pairs,
		num_keyval_pairs_after_combiner,
		(num_keyval_pairs_after_combiner/(float)total_intermediate_keyvalue_pairs)
		);
	*/
	return NULL;
}

void RunGPUMapTasksHost(panda_gpu_context pgc, int curIter, int totalIter, dim3 grids, dim3 blocks){
	PandaRunGPUMapTasks<<<grids,blocks>>>(pgc, totalIter -1 - curIter, totalIter);
	hipDeviceSynchronize();
}//void

__global__ void GPUCombiner(panda_gpu_context pgc);

void ExecutePandaGPUCombiner(panda_gpu_context * pgc){

	hipMemset(pgc->intermediate_key_vals.d_intermediate_keyval_total_count,0,pgc->input_key_vals.num_input_record*sizeof(int));
	ShowLog("pgc->input_key_vals.num_input_record:%d",pgc->input_key_vals.num_input_record);
	int numGPUCores = getGPUCoresNum();
	dim3 blocks(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
	int numBlocks = (numGPUCores*16+(blocks.x*blocks.y)-1)/(blocks.x*blocks.y);
    	dim3 grids(numBlocks, 1);

	GPUCombiner<<<grids,blocks>>>(*pgc);

	hipDeviceSynchronize();
}

void ExecutePandaCPUCombiner(panda_cpu_context *pcc){

	if (pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p == NULL)	{ ShowError("intermediate_keyval_arr_arr_p == NULL"); exit(-1); }
	if (pcc->intermediate_key_vals.intermediate_keyval_arr_arr_len <= 0)	{ ShowError("no any input keys"); exit(-1); }
	if (pcc->num_cpus_cores <= 0)	{ ShowError("pcc->num_cpus == 0"); exit(-1); }

	//-------------------------------------------------------
	//1, prepare buffer to store intermediate results
	//-------------------------------------------------------

	//keyval_arr_t *d_keyval_arr_p;

	ShowLog("num_input_record:%d",pcc->input_key_vals.num_input_record);

	int num_threads = pcc->num_cpus_cores > pcc->input_key_vals.num_input_record ? pcc->input_key_vals.num_input_record : pcc->num_cpus_cores;
	int num_records_per_thread = (pcc->input_key_vals.num_input_record)/(num_threads);
	int start_task_idx = 0;
	int end_task_idx = 0;

	for (int tid = 0;tid<num_threads;tid++){
		end_task_idx = start_task_idx + num_records_per_thread;
		if (tid < (pcc->input_key_vals.num_input_record % num_threads) )
			end_task_idx++;
		if (end_task_idx > pcc->input_key_vals.num_input_record)
			end_task_idx = pcc->input_key_vals.num_input_record;

		pcc->panda_cpu_task_thread_info[tid].start_row_idx	= start_task_idx;
		pcc->panda_cpu_task_thread_info[tid].end_row_idx	= end_task_idx;
		
		if (pthread_create(&(pcc->panda_cpu_task_thread[tid]),NULL,RunPandaCPUCombinerThread,(char *)&(pcc->panda_cpu_task_thread_info[tid]))!=0) 
			ShowError("Thread creation failed!");
		start_task_idx = end_task_idx;
	}//for
	
	for (int tid = 0; tid<num_threads; tid++){
		void *exitstat;
		if (pthread_join(pcc->panda_cpu_task_thread[tid],&exitstat)!=0) ShowError("joining failed");
	}//for
}//void


void ExecutePandaSortBucket(panda_node_context *pnc)
{

	  int numRecvedBuckets = pnc->recv_buckets.counts.size();
	  ShowLog("numRecvedBuckets:%d",numRecvedBuckets);

	  keyvals_t *sorted_intermediate_keyvals_arr = NULL; 
          pnc->sorted_key_vals.sorted_intermediate_keyvals_arr = NULL;
          pnc->sorted_key_vals.sorted_keyvals_arr_len = 0;

	  char *key_0, *key_1;
	  int keySize_0, keySize_1;
	  char *val_0;
	  // char *val_1;
	  int valSize_0;
	  //int valSize_1;
	  for(int i=0; i<numRecvedBuckets; i++){
		char *keyBuff = pnc->recv_buckets.savedKeysBuff[i];
		char *valBuff = pnc->recv_buckets.savedValsBuff[i];
		int *counts = pnc->recv_buckets.counts[i];

		int *keyPosArray  = pnc->recv_buckets.keyPos[i];
		int *keySizeArray = pnc->recv_buckets.keySize[i];
		int *valPosArray  = pnc->recv_buckets.valPos[i];
		int *valSizeArray = pnc->recv_buckets.valSize[i];

		int maxlen	= counts[0];
		int keyBuffSize	= counts[1];
		int valBuffSize	= counts[2];

		for (int j=0; j<maxlen; j++){
			
			if( keyPosArray[j] + keySizeArray[j] > keyBuffSize ) 
				ShowError("(keyPosArray[j]:%d + keySizeArray[j]:%d > keyBuffSize:%d)", keyPosArray[j], keySizeArray[j] , keyBuffSize);

			key_0		= keyBuff + keyPosArray[j];
			keySize_0	= keySizeArray[j];

			int k = 0;
			for ( ;k<pnc->sorted_key_vals.sorted_keyvals_arr_len;k++){

				key_1		= (char *)(sorted_intermediate_keyvals_arr[k].key);
				keySize_1	= sorted_intermediate_keyvals_arr[k].keySize;
				if(panda_cpu_compare(key_0,keySize_0,key_1,keySize_1)!=0)
				continue;
				//ShowLog("ExecutePandaSortBucket j:[%d] k:[%d]   key_0:[%s] key_1:[%s]",j,k,key_0,key_1);
				val_t *vals = sorted_intermediate_keyvals_arr[k].vals;
				int index   = sorted_intermediate_keyvals_arr[k].val_arr_len;
				
				sorted_intermediate_keyvals_arr[k].val_arr_len++;
				sorted_intermediate_keyvals_arr[k].vals = (val_t*)realloc(vals, sizeof(val_t)*(sorted_intermediate_keyvals_arr[k].val_arr_len));
				
				val_0   = valBuff + valPosArray[j];
				valSize_0 = valSizeArray[j];

				sorted_intermediate_keyvals_arr[k].vals[index].val = (char *)malloc(sizeof(char)*valSize_0);
				sorted_intermediate_keyvals_arr[k].vals[index].valSize = valSize_0;
				memcpy(sorted_intermediate_keyvals_arr[k].vals[index].val, val_0, valSize_0);
			
				break;
			}//for k
			if (k == pnc->sorted_key_vals.sorted_keyvals_arr_len){

			if (pnc->sorted_key_vals.sorted_keyvals_arr_len == 0) sorted_intermediate_keyvals_arr = NULL;

			int index = pnc->sorted_key_vals.sorted_keyvals_arr_len;
			pnc->sorted_key_vals.sorted_keyvals_arr_len++;
			sorted_intermediate_keyvals_arr = (keyvals_t *)realloc(sorted_intermediate_keyvals_arr, 
				sizeof(keyvals_t)*(pnc->sorted_key_vals.sorted_keyvals_arr_len));
			keyvals_t* kvalsp = (keyvals_t *)&(sorted_intermediate_keyvals_arr[index]);

			kvalsp->keySize = keySize_0;
			kvalsp->key = malloc(sizeof(char)*keySize_0);
			memcpy(kvalsp->key, key_0, keySize_0);

			kvalsp->vals = (val_t *)malloc(sizeof(val_t)*1);
			kvalsp->val_arr_len = 1;

			if (valPosArray[j] + valSizeArray[j] > valBuffSize)
				ShowError("(valPosArray[j] + valSizeArray[j] > valBuffSize)");

			val_0   = valBuff + valPosArray[j];
			valSize_0 = valSizeArray[j];

			kvalsp->vals[0].valSize = valSize_0;
			kvalsp->vals[0].val = (char *)malloc(sizeof(char)*valSize_0);
			memcpy(kvalsp->vals[0].val, val_0, valSize_0);

			}//k
		}//j
	  }//i
	  pnc->sorted_key_vals.sorted_intermediate_keyvals_arr = sorted_intermediate_keyvals_arr;
}//			



panda_gpu_context *CreatePandaGPUContext(){
	
	panda_gpu_context *pgc = (panda_gpu_context*)malloc(sizeof(panda_gpu_context));
	if (pgc == NULL) exit(-1);
	memset(pgc, 0, sizeof(panda_gpu_context));
	
	pgc->input_key_vals.d_input_keys_shared_buff = NULL;
	pgc->input_key_vals.d_input_keyval_arr = NULL;
	pgc->input_key_vals.d_input_keyval_pos_arr = NULL;
	pgc->input_key_vals.d_input_vals_shared_buff = NULL;
	pgc->input_key_vals.h_input_keyval_arr = NULL;
	pgc->input_key_vals.num_input_record = 0;
	
	pgc->intermediate_key_vals.d_intermediate_keys_shared_buff = NULL;
	pgc->intermediate_key_vals.d_intermediate_keyval_arr = NULL;
	pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_len = 0;
	pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_p = NULL;
	pgc->intermediate_key_vals.d_intermediate_keyval_pos_arr = NULL;
	pgc->intermediate_key_vals.d_intermediate_keyval_total_count = 0;
	
	pgc->sorted_key_vals.d_sorted_keyvals_arr_len = 0;
	pgc->reduced_key_vals.d_reduced_keyval_arr_len = 0;
	
	return pgc;
}//gpu_context


panda_cpu_context *CreatePandaCPUContext(){
	
	panda_cpu_context *pcc = (panda_cpu_context*)malloc(sizeof(panda_cpu_context));
	if (pcc == NULL) exit(-1);
	memset(pcc, 0, sizeof(panda_cpu_context));
	
	pcc->input_key_vals.num_input_record = 0;
	pcc->input_key_vals.input_keys_shared_buff = NULL;
	pcc->input_key_vals.input_keyval_arr = NULL;
	pcc->input_key_vals.input_keyval_pos_arr = NULL;
	pcc->input_key_vals.input_vals_shared_buff = NULL;
	pcc->input_key_vals.input_keyval_arr = NULL;
	
	pcc->intermediate_key_vals.intermediate_keys_shared_buff = NULL;
	pcc->intermediate_key_vals.intermediate_keyval_arr = NULL;
	pcc->intermediate_key_vals.intermediate_keyval_arr_arr_len = 0;
	pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p = NULL;
	pcc->intermediate_key_vals.intermediate_keyval_pos_arr = NULL;
	pcc->intermediate_key_vals.intermediate_keyval_total_count = NULL;
	
	pcc->sorted_key_vals.sorted_keyvals_arr_len = 0;
	pcc->reduced_key_vals.reduced_keyval_arr_len = 0;
	return pcc;
	
}//gpu_context

void ExecutePandaCPUReduceTasks(panda_cpu_context *pcc){

	//panda_cpu_context *pcc = this->pCPUContext;
	
	if (pcc->sorted_key_vals.sorted_keyvals_arr_len <= 0) return;
	
	for (int map_idx = 0; map_idx < pcc->sorted_key_vals.sorted_keyvals_arr_len; map_idx++){
	
		keyvals_t *kv_p = (keyvals_t *)(&(pcc->sorted_key_vals.sorted_intermediate_keyvals_arr[map_idx]));

		if (kv_p->val_arr_len <=0) 
			ShowError("kv_p->val_arr_len <=0");
		else	
			panda_cpu_reduce(kv_p->key, kv_p->vals, kv_p->keySize, kv_p->val_arr_len, pcc);
	}//for
		
}//void



void PandaExecuteSortBucketOnCPU(panda_node_context *pnc)
{
	  int numBucket = pnc->recv_buckets.savedKeysBuff.size();
	  keyvals_t *sorted_intermediate_keyvals_arr = pnc->sorted_key_vals.sorted_intermediate_keyvals_arr;
	  char *key_0, *key_1;
	  int keySize_0, keySize_1;
	  char *val_0; 	
	  // char *val_1;
	  int valSize_0; 
	  // int valSize_1;

	  //bool equal;
	  for(int i=0; i<numBucket; i++){
			
		char *keyBuff = pnc->recv_buckets.savedKeysBuff[i];
		char *valBuff = pnc->recv_buckets.savedValsBuff[i];
		int *counts = pnc->recv_buckets.counts[i];

		int *keyPosArray  = pnc->recv_buckets.keyPos[i];
		int *keySizeArray = pnc->recv_buckets.keySize[i];
		int *valPosArray  = pnc->recv_buckets.valPos[i];
		int *valSizeArray = pnc->recv_buckets.valSize[i];

		int maxlen		= counts[0];
		int keyBuffSize	= counts[1];
		int valBuffSize	= counts[2];

		for (int j=0; j<maxlen; j++){
			
			if( keyPosArray[j] + keySizeArray[j] > keyBuffSize ) 
				ShowError("keyPosArray[j]:%d + keySizeArray[j]:%d > keyBuffSize:%d", keyPosArray[j], keySizeArray[j] , keyBuffSize);

			key_0		= keyBuff + keyPosArray[j];
			keySize_0	= keySizeArray[j];

			int k = 0;
			for ( ; k < pnc->sorted_key_vals.sorted_keyvals_arr_len; k++){

				key_1		= (char *)(sorted_intermediate_keyvals_arr[k].key);
				keySize_1	= sorted_intermediate_keyvals_arr[k].keySize;

				if(panda_cpu_compare(key_0,keySize_0,key_1,keySize_1)!=0)
					continue;

				val_t *vals = sorted_intermediate_keyvals_arr[k].vals;
				int index   = sorted_intermediate_keyvals_arr[k].val_arr_len;
				
				sorted_intermediate_keyvals_arr[k].val_arr_len++;
				sorted_intermediate_keyvals_arr[k].vals = (val_t*)realloc(vals, sizeof(val_t)*(sorted_intermediate_keyvals_arr[k].val_arr_len));
				
				val_0   = valBuff + valPosArray[j];
				valSize_0 = valSizeArray[j];

				sorted_intermediate_keyvals_arr[k].vals[index].val = (char *)malloc(sizeof(char)*valSize_0);
				sorted_intermediate_keyvals_arr[k].vals[index].valSize = valSize_0;
				memcpy(sorted_intermediate_keyvals_arr[k].vals[index].val, val_0, valSize_0);
				break;
			}//for k

			if (k == pnc->sorted_key_vals.sorted_keyvals_arr_len){

			if (pnc->sorted_key_vals.sorted_keyvals_arr_len == 0) sorted_intermediate_keyvals_arr = NULL;

			int index = pnc->sorted_key_vals.sorted_keyvals_arr_len;
			pnc->sorted_key_vals.sorted_keyvals_arr_len++;
			sorted_intermediate_keyvals_arr = (keyvals_t *)realloc(sorted_intermediate_keyvals_arr, sizeof(keyvals_t)*(pnc->sorted_key_vals.sorted_keyvals_arr_len));
			
			keyvals_t* kvals_p = (keyvals_t *)&(sorted_intermediate_keyvals_arr[index]);

			kvals_p->keySize = keySize_0;
			kvals_p->key = malloc(sizeof(char)*keySize_0);
			memcpy(kvals_p->key, key_0, keySize_0);

			kvals_p->vals = (val_t *)malloc(sizeof(val_t)*1);
			kvals_p->val_arr_len = 1;

			if (valPosArray[j] + valSizeArray[j] > valBuffSize) ShowError("valPosArray[j] + valSizeArray[j] > valBuffSize");

			val_0   = valBuff + valPosArray[j];
			valSize_0 = valSizeArray[j];

			kvals_p->vals[k].valSize = valSize_0;
			kvals_p->vals[k].val = (char *)malloc(sizeof(char)*valSize_0);
			memcpy(kvals_p->vals[k].val, val_0, valSize_0);

			}//k
		}//j
	  }//i
	  pnc->sorted_key_vals.sorted_intermediate_keyvals_arr = sorted_intermediate_keyvals_arr;
}//			
			
void AddReduceTaskOnCPU(panda_cpu_context* pcc, panda_node_context *pnc, int start_task_id, int end_task_id){
		
    if (end_task_id <= start_task_id)
	{
		ShowError("end_task_id:%d<=start_task_id:%d Warning!",end_task_id,start_task_id);
		pcc->sorted_key_vals.sorted_keyvals_arr_len = 0;
		return;
	}//if

	int len = pnc->sorted_key_vals.sorted_keyvals_arr_len;
	if (len < (end_task_id - start_task_id) )
	{
		ShowError("error! pnc->sorted_key_vals.sorted_keyvals_arr_len < pcc->sorted_key_vals.sorted_keyvals_arr_len");
		pcc->sorted_key_vals.sorted_keyvals_arr_len = 0;
		return;
	}

	if (len == 0) {
		ShowError("error! pnc->sorted_key_vals.sorted_keyvals_arr_len = 0");
		pcc->sorted_key_vals.sorted_keyvals_arr_len = 0;
		return;
	}

	pcc->sorted_key_vals.sorted_intermediate_keyvals_arr = (keyvals_t *)malloc(sizeof(keyvals_t)*(end_task_id - start_task_id));
	pcc->sorted_key_vals.totalKeySize = pnc->sorted_key_vals.totalKeySize;
	pcc->sorted_key_vals.totalValSize = pnc->sorted_key_vals.totalValSize;
	
	for (int i = 0; i< end_task_id - start_task_id; i++){
	pcc->sorted_key_vals.sorted_intermediate_keyvals_arr[i].keySize		= pnc->sorted_key_vals.sorted_intermediate_keyvals_arr[start_task_id+i].keySize;
	pcc->sorted_key_vals.sorted_intermediate_keyvals_arr[i].key		= pnc->sorted_key_vals.sorted_intermediate_keyvals_arr[start_task_id+i].key;
	pcc->sorted_key_vals.sorted_intermediate_keyvals_arr[i].vals		= pnc->sorted_key_vals.sorted_intermediate_keyvals_arr[start_task_id+i].vals;
	pcc->sorted_key_vals.sorted_intermediate_keyvals_arr[i].val_arr_len = pnc->sorted_key_vals.sorted_intermediate_keyvals_arr[start_task_id+i].val_arr_len;
	}//for
	pcc->sorted_key_vals.sorted_keyvals_arr_len = end_task_id - start_task_id;

}//void AddReduceTaskOnCPU


void AddReduceTaskOnGPU(panda_gpu_context* pgc, panda_node_context *pnc, int start_task_id, int end_task_id){
	
	//end_task_id = pnc->sorted_key_vals.sorted_keyvals_arr_len;
	
	if(end_task_id <= start_task_id)
		return;
		
	keyvals_t * sorted_intermediate_keyvals_arr = pnc->sorted_key_vals.sorted_intermediate_keyvals_arr;
		
	int total_count = 0;
	for(int i=start_task_id;i<end_task_id;i++){
		total_count += sorted_intermediate_keyvals_arr[i].val_arr_len;
	}//for

	//ShowLog("start_task_id:%d end_task_id:%d  total_count:%d", start_task_id, end_task_id, total_count);
		
	int totalKeySize = 0;
	int totalValSize = 0;
	for(int i=start_task_id;i<end_task_id;i++){
		totalKeySize += (sorted_intermediate_keyvals_arr[i].keySize+3)/4*4;
		for (int j=0;j<sorted_intermediate_keyvals_arr[i].val_arr_len;j++)
		totalValSize += (sorted_intermediate_keyvals_arr[i].vals[j].valSize+3)/4*4;
	}//for
	
	ShowLog("start_task_id:%d end_task_id:%d totalKeySize:%d totalValSize:%d  total_count:%d", 
		start_task_id, end_task_id, totalKeySize, totalValSize, total_count);

	hipMalloc((void **)&(pgc->sorted_key_vals.d_sorted_keys_shared_buff), totalKeySize);
	hipMalloc((void **)&(pgc->sorted_key_vals.d_sorted_vals_shared_buff), totalValSize);
	hipMalloc((void **)&(pgc->sorted_key_vals.d_keyval_pos_arr), sizeof(keyval_pos_t)*total_count);

	pgc->sorted_key_vals.h_sorted_keys_shared_buff = malloc(sizeof(char)*totalKeySize);
	pgc->sorted_key_vals.h_sorted_vals_shared_buff = malloc(sizeof(char)*totalValSize);
		
	char *sorted_keys_shared_buff = (char *)pgc->sorted_key_vals.h_sorted_keys_shared_buff;
	char *sorted_vals_shared_buff = (char *)pgc->sorted_key_vals.h_sorted_vals_shared_buff;
	char *keyval_pos_arr = (char *)malloc(sizeof(keyval_pos_t)*total_count);

	ShowLog("start_task_id:%d end_task_id:%d  total_count:%d", start_task_id, end_task_id, total_count);
		
	int sorted_key_arr_len = end_task_id - start_task_id;
	keyval_pos_t *tmp_keyval_pos_arr = (keyval_pos_t *)malloc(sizeof(keyval_pos_t)*total_count);
		
	//ShowLog("GPU_ID:[%d] total #different intermediate records:%d total records:%d totalKeySize:%d KB totalValSize:%d KB", 
	//	d_g_state->gpu_id, end_task_id - start_task_id, total_count, totalKeySize/1024, totalValSize/1024);
		
	int *pos_arr_4_pos_arr = (int*)malloc(sizeof(int)*(sorted_key_arr_len));
	memset(pos_arr_4_pos_arr,0,sizeof(int)*sorted_key_arr_len);
		
	int index = 0;
	int keyPos = 0;
	int valPos = 0;
		
	for (int i = start_task_id; i < end_task_id; i++){
		
		keyvals_t* p = (keyvals_t*)&(sorted_intermediate_keyvals_arr[i]);
		memcpy(sorted_keys_shared_buff+keyPos,p->key, p->keySize);
		
		for (int j=0;j<p->val_arr_len;j++){
			tmp_keyval_pos_arr[index].keyPos = keyPos;
			tmp_keyval_pos_arr[index].keySize = p->keySize;
			tmp_keyval_pos_arr[index].valPos = valPos;
			tmp_keyval_pos_arr[index].valSize = p->vals[j].valSize;
			memcpy(sorted_vals_shared_buff + valPos,p->vals[j].val,p->vals[j].valSize);
			valPos += (p->vals[j].valSize+3)/4*4;
			index++;
		}//for
		keyPos += (p->keySize+3)/4*4;
		pos_arr_4_pos_arr[i-start_task_id] = index;
	}//	
		
	pgc->sorted_key_vals.d_sorted_keyvals_arr_len = end_task_id-start_task_id;
	hipMemcpy(pgc->sorted_key_vals.d_keyval_pos_arr,tmp_keyval_pos_arr,sizeof(keyval_pos_t)*total_count,hipMemcpyHostToDevice);
	hipMalloc((void**)&(pgc->sorted_key_vals.d_pos_arr_4_sorted_keyval_pos_arr),sizeof(int)*sorted_key_arr_len);
	hipMemcpy(pgc->sorted_key_vals.d_pos_arr_4_sorted_keyval_pos_arr,pos_arr_4_pos_arr,sizeof(int)*sorted_key_arr_len,hipMemcpyHostToDevice);
	hipMemcpy(pgc->sorted_key_vals.d_sorted_keys_shared_buff, sorted_keys_shared_buff, sizeof(char)*totalKeySize,hipMemcpyHostToDevice);
	hipMemcpy(pgc->sorted_key_vals.d_sorted_vals_shared_buff, sorted_vals_shared_buff, sizeof(char)*totalValSize,hipMemcpyHostToDevice);

}

__global__ void copyDataFromDevice2Host4Reduce(panda_gpu_context pgc)
{

        int num_records_per_thread = (pgc.reduced_key_vals.d_reduced_keyval_arr_len
                + (gridDim.x*blockDim.x*blockDim.y)-1)/(gridDim.x*blockDim.x*blockDim.y);
        int block_start_idx = num_records_per_thread * blockIdx.x * blockDim.x * blockDim.y;
        int thread_start_idx = block_start_idx
                + (threadIdx.y*blockDim.x + threadIdx.x)*num_records_per_thread;
        int thread_end_idx = thread_start_idx + num_records_per_thread;
        
	if(thread_end_idx> pgc.reduced_key_vals.d_reduced_keyval_arr_len)
                thread_end_idx = pgc.reduced_key_vals.d_reduced_keyval_arr_len;
        if (thread_start_idx >= thread_end_idx)
                return;

        int val_pos=0, key_pos=0;
        for (int i=0; i<thread_start_idx; i++){
                val_pos += (pgc.reduced_key_vals.d_reduced_keyval_arr[i].valSize+3)/4*4;
                key_pos += (pgc.reduced_key_vals.d_reduced_keyval_arr[i].keySize+3)/4*4;
        }//for
	
        for (int i = thread_start_idx; i < thread_end_idx;i++){
                memcpy( (char *)(pgc.output_key_vals.d_KeyBuff) + key_pos,
                        (char *)(pgc.reduced_key_vals.d_reduced_keyval_arr[i].key), pgc.reduced_key_vals.d_reduced_keyval_arr[i].keySize);
                //key_pos += pgc.reduced_key_vals.d_reduced_keyval_arr[i].keySize;
                memcpy( (char *)(pgc.output_key_vals.d_ValBuff) + val_pos,
                        (char *)(pgc.reduced_key_vals.d_reduced_keyval_arr[i].val), pgc.reduced_key_vals.d_reduced_keyval_arr[i].valSize);
       		printf("[copyDataFromDevice2Host4Reduce] key:[%s] val:[%d]\n",(char *)pgc.output_key_vals.d_KeyBuff+key_pos,
					*(int *)pgc.output_key_vals.d_ValBuff+val_pos);
        
		key_pos += pgc.reduced_key_vals.d_reduced_keyval_arr[i].keySize;
		val_pos += pgc.reduced_key_vals.d_reduced_keyval_arr[i].valSize;
  
	 }//for
	
}//__global__

void PandaEmitCPUMapOutput(void *key, void *val, int keySize, int valSize, panda_cpu_context *pcc, int map_task_idx){
	
	if(map_task_idx >= pcc->input_key_vals.num_input_record) {	ShowError("error ! map_task_idx >= d_g_state->num_input_record");		return;	}
	keyval_arr_t *kv_arr_p = &(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_task_idx]);

	char *buff = (char*)(kv_arr_p->shared_buff);
	
	if (!((*kv_arr_p->shared_buff_pos) + keySize + valSize < (*kv_arr_p->shared_buff_len) - sizeof(keyval_pos_t)*((*kv_arr_p->shared_arr_len)+1))){
		ShowWarn("Warning! not enough memory at CPU task:%d *kv_arr_p->shared_arr_len:%d current buff_size:%d KB",
			map_task_idx,*kv_arr_p->shared_arr_len,(*kv_arr_p->shared_buff_len)/1024);

		char *new_buff = (char*)malloc(sizeof(char)*((*kv_arr_p->shared_buff_len)*2));
		if(new_buff==NULL){ ShowError("Error ! There is not enough memory to allocat!"); return; }

		memcpy(new_buff, buff, sizeof(char)*(*kv_arr_p->shared_buff_pos));
		int blockSize = sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len);
		memcpy(new_buff + (*kv_arr_p->shared_buff_len)*2 - blockSize, 
			(char*)buff + (*kv_arr_p->shared_buff_len) - blockSize,
														blockSize);
		
		(*kv_arr_p->shared_buff_len) = 2*(*kv_arr_p->shared_buff_len);
		for(int  idx = 0; idx < (kv_arr_p->shared_buddy_len); idx++){
			int cur_map_task_idx = kv_arr_p->shared_buddy[idx];  //the buddy relationship won't be changed
			keyval_arr_t *cur_kv_arr_p = &(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[cur_map_task_idx]);
			cur_kv_arr_p->shared_buff = new_buff;
		}//for
		free(buff);//
		buff = new_buff;
	}//if
	
	keyval_pos_t *kv_p = (keyval_pos_t *)((char *)buff + *kv_arr_p->shared_buff_len - sizeof(keyval_pos_t)*((*kv_arr_p->shared_arr_len)+1));
	(*kv_arr_p->shared_arr_len)++;
	kv_p->task_idx = map_task_idx;
	kv_p->next_idx = _MAP;
	
	kv_p->keyPos = (*kv_arr_p->shared_buff_pos);
	*kv_arr_p->shared_buff_pos += ((keySize+3)/4)*4;		//alignment 4 bytes for reading and writing
	memcpy((char *)(buff) + kv_p->keyPos, key, keySize);
	kv_p->keySize = keySize;
	
	kv_p->valPos = (*kv_arr_p->shared_buff_pos);
	*kv_arr_p->shared_buff_pos += ((valSize+3)/4)*4;
	//char *val_p = (char *)(buff) + kv_p->valPos;
	memcpy((char *)(buff) + kv_p->valPos, val, valSize);
	kv_p->valSize = valSize;
	(kv_arr_p->arr) = kv_p;

}//

void PandaCPUEmitReduceOutput (	void*		key,
				void*		val,
				int		keySize,
				int		valSize,
				panda_cpu_context *pcc){
	/*
			keyval_t *p = (keyval_t *)(&(pcc->reduced_key_vals.reduced_keyval_arr[0]));
			p->keySize = keySize;
			p->key = malloc(keySize);
			memcpy(p->key,key,keySize);
			p->valSize = valSize;
			p->val = malloc(valSize);
			memcpy(p->val,val,valSize);
	*/
			ShowLog("[panda_reduce_output]:key:%s  val:%d",(char*)key,*(int *)val);

}

__device__ void PandaGPUEmitMapOutput(void *key, void *val, int keySize, int valSize, panda_gpu_context *pgc, int map_task_idx){
	
	keyval_arr_t *kv_arr_p = pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_p[map_task_idx];
	char *buff = (char*)(kv_arr_p->shared_buff);
	
	int shared_buff_len		= *kv_arr_p->shared_buff_len;
	int shared_arr_len		= *kv_arr_p->shared_arr_len;
	int shared_buff_pos		= *kv_arr_p->shared_buff_pos;

	int required_mem_len	= (shared_buff_pos) + keySize + valSize + sizeof(keyval_pos_t)*(shared_arr_len+1);
	//if (!((*kv_arr_p->shared_buff_pos) + keySize + valSize <    - sizeof(keyval_pos_t)*((*kv_arr_p->shared_arr_len)+1))){

	if (required_mem_len > shared_buff_len){

		while (required_mem_len >= shared_buff_len){
			shared_buff_len *= 2;
		}//while
		
		ShowWarn("Warning! not enough memory at GPU task:%d *kv_arr_p->shared_arr_len:%d current buff_size:%d KB",
			map_task_idx,*kv_arr_p->shared_arr_len,(*kv_arr_p->shared_buff_len)/1024);
		
		char *new_buff = (char*)malloc(sizeof(char)*(shared_buff_len));
		if(new_buff==NULL){ ShowWarn("Error ! There is not enough memory to allocat!"); return; }
		
		memcpy(new_buff, buff, sizeof(char)*(*kv_arr_p->shared_buff_pos));
		memcpy(new_buff + (shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len), 
			(char*)buff + (*kv_arr_p->shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len),
														sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len));
				
		(*kv_arr_p->shared_buff_len) = (shared_buff_len);
				
		for(int  idx = 0; idx < (kv_arr_p->shared_buddy_len); idx++){
				
			int cur_map_task_idx = kv_arr_p->shared_buddy[idx];  //the buddy relationship won't be changed 
			keyval_arr_t *cur_kv_arr_p = pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_p[cur_map_task_idx];
			cur_kv_arr_p->shared_buff = new_buff;
				
		}//for
		free(buff);//?????
		buff = new_buff;
	}//if
	
	keyval_pos_t *kv_p = (keyval_pos_t *)((char *)buff + *kv_arr_p->shared_buff_len - sizeof(keyval_pos_t)*((*kv_arr_p->shared_arr_len)+1));
	(*kv_arr_p->shared_arr_len)++;
	kv_p->task_idx = map_task_idx;
	kv_p->next_idx = _MAP;

	kv_p->keyPos = (*kv_arr_p->shared_buff_pos);
	*kv_arr_p->shared_buff_pos += ((keySize+3)/4)*4;		//alignment 4 bytes for reading and writing
	memcpy((char *)(buff) + kv_p->keyPos,key,keySize);
	kv_p->keySize = keySize;
	
	kv_p->valPos = (*kv_arr_p->shared_buff_pos);
	*kv_arr_p->shared_buff_pos += ((valSize+3)/4)*4;
	//char *val_p = (char *)(buff) + kv_p->valPos;
	memcpy((char *)(buff) + kv_p->valPos, val, valSize);
	kv_p->valSize = valSize;
	(kv_arr_p->arr) = kv_p;

	kv_arr_p->arr_len++;
	pgc->intermediate_key_vals.d_intermediate_keyval_total_count[map_task_idx] = kv_arr_p->arr_len;

}//__device__

__device__ void PandaGPUEmitCombinerOutput(void *key, void *val, int keySize, int valSize, panda_gpu_context *pgc, int map_task_idx){
			
	keyval_arr_t *kv_arr_p	= pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_p[map_task_idx];
	void *shared_buff		= kv_arr_p->shared_buff;
	int shared_buff_len		= *kv_arr_p->shared_buff_len;
	int shared_arr_len		= *kv_arr_p->shared_arr_len;
	int shared_buff_pos		= *kv_arr_p->shared_buff_pos;
		
	int required_mem_len = (shared_buff_pos) + keySize + valSize + sizeof(keyval_pos_t)*(shared_arr_len+1);
	if (required_mem_len> shared_buff_len){

		while (required_mem_len>= shared_buff_len){
			shared_buff_len *= 2;
		}//while

		ShowWarn("Warning! no enough memory in GPU task:%d need:%d KB KeySize:%d ValSize:%d shared_arr_len:%d shared_buff_pos:%d shared_buff_len:%d",
			map_task_idx, required_mem_len/1024,keySize,valSize,shared_arr_len,shared_buff_pos,shared_buff_len);
		
		char *new_buff = (char*)malloc(sizeof(char)*(shared_buff_len));
		if(new_buff==NULL)ShowWarn(" There is not enough memory to allocat!");

		memcpy(new_buff, shared_buff, sizeof(char)*(*kv_arr_p->shared_buff_pos));
		memcpy(new_buff + (shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len), 
			(char*)shared_buff + (*kv_arr_p->shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len),
												sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len));
		
		(*kv_arr_p->shared_buff_len) = shared_buff_len;	
		
		for(int  idx = 0; idx < (kv_arr_p->shared_buddy_len); idx++){

		int cur_map_task_idx = kv_arr_p->shared_buddy[idx];  //the buddy relationship won't be changed 
		keyval_arr_t *cur_kv_arr_p = pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_p[cur_map_task_idx];
		cur_kv_arr_p->shared_buff = new_buff;
		
		}//for

		free(shared_buff);
		shared_buff = new_buff;
	
	}//if

	keyval_pos_t *kv_p = (keyval_pos_t *)((char *)shared_buff + shared_buff_len - sizeof(keyval_pos_t)*(shared_arr_len + 1));
	kv_p->keySize = keySize;
	kv_p->valSize = valSize;
	kv_p->task_idx = map_task_idx;
	kv_p->next_idx = _COMBINE;			//merged results

	memcpy( (char*)shared_buff + *kv_arr_p->shared_buff_pos, key, keySize);
	kv_p->keyPos = *kv_arr_p->shared_buff_pos;
	*kv_arr_p->shared_buff_pos += (keySize+3)/4*4;

	memcpy( (char*)shared_buff + *kv_arr_p->shared_buff_pos, val, valSize);
	kv_p->valPos = *kv_arr_p->shared_buff_pos;
	*kv_arr_p->shared_buff_pos += (valSize+3)/4*4;
	
	(*kv_arr_p->shared_arr_len)++;
			
}//__device__

__device__ void PandaGPUEmitReduceOutput(
						void*		key, 
						void*		val, 
						int		keySize, 
						int		valSize,
						panda_gpu_context *pgc){
	//printf("[PandaGPUEmitReduceOutput] key:[%s] val:[%d] TID:%d len:%d\n",(char *)key,*(int *)val,TID,
	//		pgc->reduced_key_vals.d_reduced_keyval_arr_len);
			
		        keyval_t *p = &(pgc->reduced_key_vals.d_reduced_keyval_arr[TID]);
			p->keySize = keySize;
			p->key = malloc(keySize);
			memcpy(p->key,key,keySize);
			p->valSize = valSize;
			p->val = malloc(valSize);
			memcpy(p->val,val,valSize);
	//printf("[PandaGPUEmitRedueOutput] key:[%s] val:[%d]",(char *)p->key,*(int *)p->val);

}//__device__ 


void PandaCPUEmitCombinerOutput(void *key, void *val, int keySize, int valSize, panda_cpu_context *pcc, int map_task_idx){

	keyval_arr_t *kv_arr_p	= &(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_task_idx]);
	void *shared_buff		= kv_arr_p->shared_buff;
	int shared_buff_len		= *kv_arr_p->shared_buff_len;
	int shared_arr_len		= *kv_arr_p->shared_arr_len;
	int shared_buff_pos		= *kv_arr_p->shared_buff_pos;
		
	int required_mem_len = (shared_buff_pos) + keySize + valSize + sizeof(keyval_pos_t)*(shared_arr_len+1);
	if (required_mem_len> shared_buff_len){

		while(required_mem_len> shared_buff_len){
			shared_buff_len *= 2;
		}//while

		ShowWarn("Warning! no enough memory in GPU task:%d need:%d KB KeySize:%d ValSize:%d shared_arr_len:%d shared_buff_pos:%d shared_buff_len:%d",
			map_task_idx, required_mem_len/1024,keySize,valSize,shared_arr_len,shared_buff_pos,shared_buff_len);
		
		char *new_buff = (char*)malloc(sizeof(char)*(shared_buff_len));
		if(new_buff==NULL)ShowError(" There is not enough memory to allocat!");

		memcpy(new_buff, shared_buff, sizeof(char)*(*kv_arr_p->shared_buff_pos));
		memcpy(new_buff + (shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len), 
			(char*)shared_buff + (*kv_arr_p->shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len),
												sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len));
		
		(*kv_arr_p->shared_buff_len) = shared_buff_len;	
		
		for(int  idx = 0; idx < (kv_arr_p->shared_buddy_len); idx++){

		int cur_map_task_idx = kv_arr_p->shared_buddy[idx];			//the buddy relationship won't be changed 
		keyval_arr_t *cur_kv_arr_p = &(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[cur_map_task_idx]);
		cur_kv_arr_p->shared_buff = new_buff;
		
		}//for

		free(shared_buff);
		shared_buff = new_buff;
	
	}//if

	keyval_pos_t *kv_p = (keyval_pos_t *)((char *)shared_buff + shared_buff_len - sizeof(keyval_pos_t)*(shared_arr_len + 1));
	kv_p->keySize = keySize;
	kv_p->valSize = valSize;
	kv_p->task_idx = map_task_idx;
	kv_p->next_idx = _COMBINE;				//merged results

	memcpy( (char*)shared_buff + *kv_arr_p->shared_buff_pos, key, keySize);
	kv_p->keyPos = *kv_arr_p->shared_buff_pos;
	*kv_arr_p->shared_buff_pos += (keySize+3)/4*4;

	memcpy( (char*)shared_buff + *kv_arr_p->shared_buff_pos, val, valSize);
	kv_p->valPos = *kv_arr_p->shared_buff_pos;
	*kv_arr_p->shared_buff_pos += (valSize+3)/4*4;
	
	(*kv_arr_p->shared_arr_len)++;

}//void


__device__ void PandaEmitReduceOutputOnGPU(
						void*		key, 
						void*		val, 
						int		keySize, 
						int		valSize,
						panda_gpu_context *pgc){
						
		    keyval_t *p = &(pgc->reduced_key_vals.d_reduced_keyval_arr[TID]);
			p->keySize = keySize;
			p->key = malloc(keySize);
			memcpy(p->key,key,keySize);
			p->valSize = valSize;
			p->val = malloc(valSize);
			memcpy(p->val,val,valSize);

}//__device__ 

void PandaEmitReduceOutputOnCPU (void*	key,
				void*		val,
				int		keySize,
				int		valSize,
				panda_cpu_context *pcc){

			/*keyval_t *p = &(pcc->reduced_key_vals.reduced_keyval_arr[TID]);
			p->keySize = keySize;
			p->key = malloc(keySize);
			memcpy(p->key,key,keySize);
			p->valSize = valSize;
			p->val = malloc(valSize);
			memcpy(p->val,val,valSize);*/
			ShowLog("[panda_cpu_output]:key:%s  val:%d",(char*)key,*(int *)val);

}


__device__ void PandaEmitCombinerOutputOnGPU(void *key, void *val, int keySize, int valSize, panda_gpu_context *pgc, int map_task_idx){
			
	keyval_arr_t *kv_arr_p	= pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_p[map_task_idx];
	void *shared_buff		= kv_arr_p->shared_buff;
	int shared_buff_len		= *kv_arr_p->shared_buff_len;
	int shared_arr_len		= *kv_arr_p->shared_arr_len;
	int shared_buff_pos		= *kv_arr_p->shared_buff_pos;
		
	int required_mem_len = (shared_buff_pos) + keySize + valSize + sizeof(keyval_pos_t)*(shared_arr_len+1);
	if (required_mem_len> shared_buff_len){

		while (required_mem_len>= shared_buff_len){
			shared_buff_len *= 2;
		}//while

		ShowWarn("Warning! no enough memory in GPU task:%d need:%d KB KeySize:%d ValSize:%d shared_arr_len:%d shared_buff_pos:%d shared_buff_len:%d",
			map_task_idx, required_mem_len/1024,keySize,valSize,shared_arr_len,shared_buff_pos,shared_buff_len);
		
		char *new_buff = (char*)malloc(sizeof(char)*(shared_buff_len));
		if(new_buff==NULL)ShowWarn(" There is not enough memory to allocat!");

		memcpy(new_buff, shared_buff, sizeof(char)*(*kv_arr_p->shared_buff_pos));
		memcpy(new_buff + (shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len), 
			(char*)shared_buff + (*kv_arr_p->shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len),
												sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len));
		
		(*kv_arr_p->shared_buff_len) = shared_buff_len;	
		
		for(int  idx = 0; idx < (kv_arr_p->shared_buddy_len); idx++){

		int cur_map_task_idx = kv_arr_p->shared_buddy[idx];  //the buddy relationship won't be changed 
		keyval_arr_t *cur_kv_arr_p = pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_p[cur_map_task_idx];
		cur_kv_arr_p->shared_buff = new_buff;
		
		}//for

		free(shared_buff);
		shared_buff = new_buff;
	
	}//if

	keyval_pos_t *kv_p = (keyval_pos_t *)((char *)shared_buff + shared_buff_len - sizeof(keyval_pos_t)*(shared_arr_len + 1));
	kv_p->keySize = keySize;
	kv_p->valSize = valSize;
	kv_p->task_idx = map_task_idx;
	kv_p->next_idx = _COMBINE;			//merged results

	memcpy( (char*)shared_buff + *kv_arr_p->shared_buff_pos, key, keySize);
	kv_p->keyPos = *kv_arr_p->shared_buff_pos;
	*kv_arr_p->shared_buff_pos += (keySize+3)/4*4;

	memcpy( (char*)shared_buff + *kv_arr_p->shared_buff_pos, val, valSize);
	kv_p->valPos = *kv_arr_p->shared_buff_pos;
	*kv_arr_p->shared_buff_pos += (valSize+3)/4*4;
	
	(*kv_arr_p->shared_arr_len)++;
			
}//__device__


__device__ void PandaEmitMapOutputOnGPU(void *key, void *val, int keySize, int valSize, panda_gpu_context *pgc, int map_task_idx){
	
	keyval_arr_t *kv_arr_p = pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_p[map_task_idx];
	char *buff = (char*)(kv_arr_p->shared_buff);
	
	int shared_buff_len		= *kv_arr_p->shared_buff_len;
	int shared_arr_len		= *kv_arr_p->shared_arr_len;
	int shared_buff_pos		= *kv_arr_p->shared_buff_pos;

	int required_mem_len	= (shared_buff_pos) + keySize + valSize + sizeof(keyval_pos_t)*(shared_arr_len+1);
	//if (!((*kv_arr_p->shared_buff_pos) + keySize + valSize <    - sizeof(keyval_pos_t)*((*kv_arr_p->shared_arr_len)+1))){

	if (required_mem_len > shared_buff_len){

		while (required_mem_len >= shared_buff_len){
			shared_buff_len *= 2;
		}//while
		
		ShowWarn("Warning! not enough memory at GPU task:%d *kv_arr_p->shared_arr_len:%d current buff_size:%d KB",
			map_task_idx,*kv_arr_p->shared_arr_len,(*kv_arr_p->shared_buff_len)/1024);
		
		char *new_buff = (char*)malloc(sizeof(char)*(shared_buff_len));
		if(new_buff==NULL){ ShowWarn("Error ! There is not enough memory to allocat!"); return; }
		
		memcpy(new_buff, buff, sizeof(char)*(*kv_arr_p->shared_buff_pos));
		memcpy(new_buff + (shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len), 
			(char*)buff + (*kv_arr_p->shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len),
														sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len));
				
		(*kv_arr_p->shared_buff_len) = (shared_buff_len);
				
		for(int  idx = 0; idx < (kv_arr_p->shared_buddy_len); idx++){
				
			int cur_map_task_idx = kv_arr_p->shared_buddy[idx];  //the buddy relationship won't be changed 
			keyval_arr_t *cur_kv_arr_p = pgc->intermediate_key_vals.d_intermediate_keyval_arr_arr_p[cur_map_task_idx];
			cur_kv_arr_p->shared_buff = new_buff;
				
		}//for
		free(buff);//?????
		buff = new_buff;
	}//if
	
	keyval_pos_t *kv_p = (keyval_pos_t *)((char *)buff + *kv_arr_p->shared_buff_len - sizeof(keyval_pos_t)*((*kv_arr_p->shared_arr_len)+1));
	(*kv_arr_p->shared_arr_len)++;
	kv_p->task_idx = map_task_idx;
	kv_p->next_idx = _MAP;

	kv_p->keyPos = (*kv_arr_p->shared_buff_pos);
	*kv_arr_p->shared_buff_pos += ((keySize+3)/4)*4;		//alignment 4 bytes for reading and writing
	memcpy((char *)(buff) + kv_p->keyPos,key,keySize);
	kv_p->keySize = keySize;
	
	kv_p->valPos = (*kv_arr_p->shared_buff_pos);
	*kv_arr_p->shared_buff_pos += ((valSize+3)/4)*4;
	//char *val_p = (char *)(buff) + kv_p->valPos;
	memcpy((char *)(buff) + kv_p->valPos, val, valSize);
	kv_p->valSize = valSize;
	(kv_arr_p->arr) = kv_p;

}//__device__


__global__ void PandaExecuteMapPartitionerOnGPU(panda_gpu_context pgc)
{

	//ShowLog("gridDim.x:%d gridDim.y:%d gridDim.z:%d blockDim.x:%d blockDim.y:%d blockDim.z:%d blockIdx.x:%d blockIdx.y:%d blockIdx.z:%d\n",
	//  gridDim.x,gridDim.y,gridDim.z,blockDim.x,blockDim.y,blockDim.z,blockIdx.x,blockIdx.y,blockIdx.z);
	int num_records_per_thread = (pgc.input_key_vals.num_input_record + (gridDim.x*blockDim.x*blockDim.y)-1)/(gridDim.x*blockDim.x*blockDim.y);
	int block_start_idx = num_records_per_thread * blockIdx.x * blockDim.x * blockDim.y;
	int thread_start_idx = block_start_idx 
		+ ((threadIdx.y*blockDim.x + threadIdx.x)/STRIDE)*num_records_per_thread*STRIDE
		+ ((threadIdx.y*blockDim.x + threadIdx.x)%STRIDE);

	int thread_end_idx = thread_start_idx + num_records_per_thread*STRIDE;
	if (thread_end_idx > pgc.input_key_vals.num_input_record)
		thread_end_idx = pgc.input_key_vals.num_input_record;

	if (thread_start_idx >= thread_end_idx)
		return;

	//if(TID==0) 	ShowWarn("hi 0 -- num_records_per_thread:%d",num_records_per_thread);

	int buddy_arr_len = num_records_per_thread;
	int * int_arr = (int*)malloc((4+buddy_arr_len)*sizeof(int));
	if(int_arr==NULL){ GpuShowError("there is not enough GPU memory\n"); return;}

	int *shared_arr_len = int_arr;
	int *shared_buff_len = int_arr+1;
	int *shared_buff_pos = int_arr+2;
	//int *num_buddy = int_arr+3;
	int *buddy = int_arr+4;
	(*shared_buff_len) = SHARED_BUFF_LEN;
	(*shared_arr_len) = 0;
	(*shared_buff_pos) = 0;

	char * buff = (char *)malloc(sizeof(char)*(*shared_buff_len));
	keyval_arr_t *kv_arr_t_arr = (keyval_arr_t *)malloc(sizeof(keyval_arr_t)*(thread_end_idx-thread_start_idx+STRIDE-1)/STRIDE);
	int index = 0;
	
	for(int idx = thread_start_idx; idx < thread_end_idx; idx += STRIDE){
			buddy[index] = idx;
			index ++;
	}//for
	index = 0;
	for(int map_task_idx = thread_start_idx; map_task_idx < thread_end_idx; map_task_idx += STRIDE){

		keyval_arr_t *kv_arr_t = (keyval_arr_t *)&(kv_arr_t_arr[index]);
		index++;
		kv_arr_t->shared_buff = buff;
		kv_arr_t->shared_arr_len = shared_arr_len;
		kv_arr_t->shared_buff_len = shared_buff_len;
		kv_arr_t->shared_buff_pos = shared_buff_pos;
		kv_arr_t->shared_buddy = buddy;
		kv_arr_t->shared_buddy_len = buddy_arr_len;
		kv_arr_t->arr = NULL;
		kv_arr_t->arr_len = 0;
		
		pgc.intermediate_key_vals.d_intermediate_keyval_arr_arr_p[map_task_idx] = kv_arr_t;

	}//for
}

void PandaLaunchMapPartitionerOnGPU(panda_gpu_context pgc, dim3 grids, dim3 blocks)
{
   PandaExecuteMapPartitionerOnGPU<<<grids,blocks>>>(pgc);
}

void PandaLaunchMapTasksOnGPU(panda_gpu_context pgc, int curIter, int totalIter, dim3 grids, dim3 blocks){
}//void



__global__ void GPUCombiner(panda_gpu_context pgc)
{

	//GpuShowError("gridDim.x:%d gridDim.y:%d gridDim.z:%d blockDim.x:%d blockDim.y:%d blockDim.z:%d blockIdx.x:%d blockIdx.y:%d blockIdx.z:%d",
	// gridDim.x,gridDim.y,gridDim.z,blockDim.x,blockDim.y,blockDim.z,blockIdx.x,blockIdx.y,blockIdx.z);

	int num_records_per_thread = (pgc.input_key_vals.num_input_record + (gridDim.x*blockDim.x*blockDim.y)-1)/(gridDim.x*blockDim.x*blockDim.y);
	//GpuShowError("num_records_per_thread:%d",num_records_per_thread);
	
	int block_start_idx = num_records_per_thread * blockIdx.x * blockDim.x * blockDim.y;
	int thread_start_idx = block_start_idx 
		+ ((threadIdx.y*blockDim.x + threadIdx.x)/STRIDE)*num_records_per_thread*STRIDE
		+ ((threadIdx.y*blockDim.x + threadIdx.x)%STRIDE);

	int thread_end_idx = thread_start_idx + num_records_per_thread;//*STRIDE;
	if (thread_end_idx > pgc.input_key_vals.num_input_record)
		thread_end_idx = pgc.input_key_vals.num_input_record;
	
	if (thread_start_idx >= thread_end_idx)
		return;

	keyval_arr_t *kv_arr_p = pgc.intermediate_key_vals.d_intermediate_keyval_arr_arr_p[thread_start_idx];
	//int *buddy = kv_arr_p->shared_buddy;

	int unmerged_shared_arr_len = *kv_arr_p->shared_arr_len;
	//GpuShowError("[GPUCombiner] unmerged_shared_arr_len:%d",unmerged_shared_arr_len);

	val_t *val_t_arr = (val_t *)malloc(sizeof(val_t)*unmerged_shared_arr_len);
	if (val_t_arr == NULL) {
	GpuShowError("[GPUCombiner] there is no enough memory. Return");
	return;
	}//if

	int num_keyval_pairs_after_combiner = 0;
	for (int i=0; i<unmerged_shared_arr_len;i++){
		
		char *shared_buff	= (kv_arr_p->shared_buff);	
		int shared_buff_len = *kv_arr_p->shared_buff_len;

		keyval_pos_t *head_kv_p = (keyval_pos_t *)(shared_buff + shared_buff_len - sizeof(keyval_pos_t)*(unmerged_shared_arr_len-i));
		keyval_pos_t *first_kv_p = head_kv_p;

		if (first_kv_p->next_idx != _MAP)
			continue;

		int iKeySize = first_kv_p->keySize;
		char *iKey = shared_buff + first_kv_p->keyPos;
		//char *iVal = shared_buff + first_kv_p->valPos;

		if((first_kv_p->keyPos%4!=0)||(first_kv_p->valPos%4!=0)){
			GpuShowError("keyPos or valPos is not aligned with 4 bytes, results could be wrong");
			return;
		}//if

		int index = 0;
		first_kv_p = head_kv_p;

		(val_t_arr[index]).valSize = first_kv_p->valSize;
		(val_t_arr[index]).val = (char*)shared_buff + first_kv_p->valPos;

		for (int j=i+1;j<unmerged_shared_arr_len;j++){

			keyval_pos_t *next_kv_p = (keyval_pos_t *)((char *)shared_buff + shared_buff_len - sizeof(keyval_pos_t)*(unmerged_shared_arr_len-j));
			char *jKey = (char *)shared_buff+next_kv_p->keyPos;
			int jKeySize = next_kv_p->keySize;
		
			if (panda_gpu_core_compare(iKey,iKeySize,jKey,jKeySize)!=0){
				continue;
			}//if
			index++;
			first_kv_p->next_idx = j;
			first_kv_p = next_kv_p;
			(val_t_arr[index]).valSize = next_kv_p->valSize;
			(val_t_arr[index]).val = (char*)shared_buff + next_kv_p->valPos;

		}//for

		int valCount = index+1;
		if(valCount>1)
		panda_gpu_core_combiner(iKey,val_t_arr,iKeySize,(valCount),&pgc,thread_start_idx);
		else{
			first_kv_p->next_idx = _COMBINE;
			first_kv_p->task_idx = thread_start_idx;
		}
		num_keyval_pairs_after_combiner++;
	}//for
	free(val_t_arr);
	pgc.intermediate_key_vals.d_intermediate_keyval_total_count[thread_start_idx] = num_keyval_pairs_after_combiner;
	__syncthreads();

}//GPU

void PandaExecuteCombinerOnCPU(panda_cpu_context *pcc){
	
	if (pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p == NULL)	{ ShowError("intermediate_keyval_arr_arr_p == NULL"); exit(-1); }
	if (pcc->intermediate_key_vals.intermediate_keyval_arr_arr_len <= 0)	{ ShowError("no any input keys"); exit(-1); }
	if (pcc->num_cpus_cores <= 0)	{ ShowError("pcc->num_cpus == 0"); exit(-1); }

	//-------------------------------------------------------
	//1, prepare buffer to store intermediate results
	//-------------------------------------------------------

	int num_threads = pcc->num_cpus_cores;
	int num_records_per_thread = (pcc->input_key_vals.num_input_record + num_threads - 1)/(num_threads);
	int start_row_idx = 0;
	int end_row_idx = 0;

	for (int tid = 0;tid<num_threads;tid++){
		end_row_idx = start_row_idx + num_records_per_thread;
		if (tid < (pcc->input_key_vals.num_input_record % num_threads) )
			end_row_idx++;
		if (end_row_idx > pcc->input_key_vals.num_input_record)
			end_row_idx = pcc->input_key_vals.num_input_record;

		pcc->panda_cpu_task_thread_info[tid].start_row_idx	= start_row_idx;
		pcc->panda_cpu_task_thread_info[tid].end_row_idx	= end_row_idx;
		
		if (pthread_create(&(pcc->panda_cpu_task_thread[tid]),NULL,PandaThreadLaunchCombinerOnCPU,(char *)&(pcc->panda_cpu_task_thread_info[tid]))!=0) 
			ShowError("Thread creation failed!");
		start_row_idx = end_row_idx;
	}//for

	for (int tid = 0; tid<num_threads; tid++){
		void *exitstat;
		if (pthread_join(pcc->panda_cpu_task_thread[tid],&exitstat)!=0) ShowError("joining failed");
	}//for

}//void


//use the region memory management technology to eliminatae the overhead of allocating and deallocating small buffers.
void *PandaThreadLaunchCombinerOnCPU(void *ptr){

	panda_cpu_task_info_t *panda_cpu_task_info = (panda_cpu_task_info_t *)ptr;
	panda_cpu_context *pcc	= (panda_cpu_task_info->pcc); 
	//panda_node_context *pnc = (panda_cpu_task_info->pnc); 

	int start_idx = panda_cpu_task_info->start_row_idx;
	int end_idx = panda_cpu_task_info->end_row_idx;

	if(start_idx>=end_idx)
		return NULL;

	//int index = 0;
	//int merged_key_arr_len = 0;
	//keyvals_t * merged_keyvals_arr = NULL;
	
	keyval_arr_t *kv_arr_p	= (keyval_arr_t *)&(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[start_idx]);

	int unmerged_shared_arr_len = *kv_arr_p->shared_arr_len;
    	//int *shared_buddy			= kv_arr_p->shared_buddy;
    	//int shared_buddy_len		= kv_arr_p->shared_buddy_len;
    	char *shared_buff = kv_arr_p->shared_buff;
    	int shared_buff_len = *kv_arr_p->shared_buff_len;
    	//int shared_buff_pos = *kv_arr_p->shared_buff_pos;

	val_t *val_t_arr = (val_t *)malloc(sizeof(val_t)*unmerged_shared_arr_len);
	if (val_t_arr == NULL) ShowError("there is no enough memory");
	int num_keyval_pairs_after_combiner = 0;
	int total_intermediate_keyvalue_pairs = 0;

	for (int i = 0; i < unmerged_shared_arr_len; i++){

		keyval_pos_t *head_kv_p = (keyval_pos_t *)(shared_buff + shared_buff_len - sizeof(keyval_pos_t)*(unmerged_shared_arr_len-i));
		keyval_pos_t *first_kv_p = head_kv_p;

		if (first_kv_p->next_idx != _MAP)
			continue;

		int iKeySize	= first_kv_p->keySize;
		char *iKey		= shared_buff + first_kv_p->keyPos;
		//char *iVal		= shared_buff + first_kv_p->valPos;

		if((first_kv_p->keyPos%4!=0)||(first_kv_p->valPos%4!=0)){
			ShowError("keyPos or valPos is not aligned with 4 bytes, results could be wrong");
		}//
	
		int index = 0;
		first_kv_p = head_kv_p;

		(val_t_arr[index]).valSize = first_kv_p->valSize;
		(val_t_arr[index]).val = (char*)shared_buff + first_kv_p->valPos;

		for (int j=i+1;j<unmerged_shared_arr_len;j++){

			keyval_pos_t *next_kv_p = (keyval_pos_t *)((char *)shared_buff + shared_buff_len - sizeof(keyval_pos_t)*(unmerged_shared_arr_len-j));
			char *jKey = (char *)shared_buff+next_kv_p->keyPos;
			int jKeySize = next_kv_p->keySize;
		
			if (panda_cpu_compare(iKey,iKeySize,jKey,jKeySize)!=0){
				continue;
			}

			index++;
			first_kv_p->next_idx = j;
			first_kv_p = next_kv_p;
			(val_t_arr[index]).valSize = next_kv_p->valSize;
			(val_t_arr[index]).val = (char*)shared_buff + next_kv_p->valPos;

		}

		int valCount = index+1;
		total_intermediate_keyvalue_pairs += valCount;
		if(valCount>1)
			panda_cpu_combiner(iKey, val_t_arr, iKeySize, (valCount), pcc, start_idx);
		else{
			first_kv_p->next_idx = _COMBINE;
			first_kv_p->task_idx = start_idx;
		}
		num_keyval_pairs_after_combiner++;
	}//for
	free(val_t_arr);
	pcc->intermediate_key_vals.intermediate_keyval_total_count[start_idx] = num_keyval_pairs_after_combiner;

	/*ShowLog("CPU_GROUP_ID:[%d] Map_Idx:%d  Done:%d Combiner: %d => %d Compress Ratio:%f",
		d_g_state->cpu_group_id, 
		panda_cpu_task_info->start_row_idx,
		panda_cpu_task_info->end_row_idx - panda_cpu_task_info->start_row_idx, 
		total_intermediate_keyvalue_pairs,
		num_keyval_pairs_after_combiner,
		(num_keyval_pairs_after_combiner/(float)total_intermediate_keyvalue_pairs)
		);*/
	return NULL;
}


//-------------------------------------------------------
//Reducer
//-------------------------------------------------------

__global__ void GPUReducePartitioner(panda_gpu_context pgc)
{
	//ShowError("ReducePartitioner Panda_GPU_Context");
	//int num_records_per_thread = (d_g_state.d_sorted_keyvals_arr_len + (gridDim.x*blockDim.x*blockDim.y)-1)/(gridDim.x*blockDim.x*blockDim.y);
	int num_records_per_thread = (pgc.sorted_key_vals.d_sorted_keyvals_arr_len + (gridDim.x*blockDim.x*blockDim.y)-1)/(gridDim.x*blockDim.x*blockDim.y);
	int block_start_idx = num_records_per_thread * blockIdx.x * blockDim.x * blockDim.y;
	int thread_start_idx = block_start_idx 
		+ ((threadIdx.y*blockDim.x + threadIdx.x)/STRIDE)*num_records_per_thread*STRIDE
		+ ((threadIdx.y*blockDim.x + threadIdx.x)%STRIDE);

	int thread_end_idx = thread_start_idx + num_records_per_thread*STRIDE;
	
	if (thread_end_idx > pgc.sorted_key_vals.d_sorted_keyvals_arr_len)
		thread_end_idx = pgc.sorted_key_vals.d_sorted_keyvals_arr_len;

	if (thread_start_idx >= thread_end_idx)
		return;

	int start_idx, end_idx;
	for(int reduce_task_idx=thread_start_idx; reduce_task_idx < thread_end_idx; reduce_task_idx+=STRIDE){

		if (reduce_task_idx==0)
			start_idx = 0;
		else
			start_idx = pgc.sorted_key_vals.d_pos_arr_4_sorted_keyval_pos_arr[reduce_task_idx-1];
		end_idx = pgc.sorted_key_vals.d_pos_arr_4_sorted_keyval_pos_arr[reduce_task_idx];
		val_t *val_t_arr = (val_t*)malloc(sizeof(val_t)*(end_idx-start_idx));
		
		int keySize = pgc.sorted_key_vals.d_keyval_pos_arr[start_idx].keySize;
		int keyPos = pgc.sorted_key_vals.d_keyval_pos_arr[start_idx].keyPos;
		void *key = (char*)pgc.sorted_key_vals.d_sorted_keys_shared_buff+keyPos;
				
		for (int index = start_idx;index<end_idx;index++){
			int valSize = pgc.sorted_key_vals.d_keyval_pos_arr[index].valSize;
			int valPos = pgc.sorted_key_vals.d_keyval_pos_arr[index].valPos;
			val_t_arr[index-start_idx].valSize = valSize;
			val_t_arr[index-start_idx].val = (char*)pgc.sorted_key_vals.d_sorted_vals_shared_buff + valPos;
		}   //for

		if( end_idx - start_idx == 0) GpuShowError("gpu_reduce valCount ==0");
		else panda_gpu_core_reduce(key, val_t_arr, keySize, end_idx-start_idx, pgc);
	}//for
}



void PandaEmitCombinerOutputOnCPU(void *key, void *val, int keySize, int valSize, panda_cpu_context *pcc, int map_task_idx){

	keyval_arr_t *kv_arr_p	= &(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_task_idx]);
	void *shared_buff		= kv_arr_p->shared_buff;
	int shared_buff_len		= *kv_arr_p->shared_buff_len;
	int shared_arr_len		= *kv_arr_p->shared_arr_len;
	int shared_buff_pos		= *kv_arr_p->shared_buff_pos;
		
	int required_mem_len = (shared_buff_pos) + keySize + valSize + sizeof(keyval_pos_t)*(shared_arr_len+1);
	if (required_mem_len> shared_buff_len){

		while(required_mem_len> shared_buff_len){
			shared_buff_len *= 2;
		}//while

		ShowWarn("Warning! no enough memory in GPU task:%d need:%d KB KeySize:%d ValSize:%d shared_arr_len:%d shared_buff_pos:%d shared_buff_len:%d",
			map_task_idx, required_mem_len/1024,keySize,valSize,shared_arr_len,shared_buff_pos,shared_buff_len);
		
		char *new_buff = (char*)malloc(sizeof(char)*(shared_buff_len));
		if(new_buff==NULL)ShowError(" There is not enough memory to allocat!");

		memcpy(new_buff, shared_buff, sizeof(char)*(*kv_arr_p->shared_buff_pos));
		memcpy(new_buff + (shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len), 
			(char*)shared_buff + (*kv_arr_p->shared_buff_len) - sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len),
												sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len));
		
		(*kv_arr_p->shared_buff_len) = shared_buff_len;	
		
		for(int  idx = 0; idx < (kv_arr_p->shared_buddy_len); idx++){

		int cur_map_task_idx = kv_arr_p->shared_buddy[idx];			//the buddy relationship won't be changed 
		keyval_arr_t *cur_kv_arr_p = &(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[cur_map_task_idx]);
		cur_kv_arr_p->shared_buff = new_buff;
		
		}//for

		free(shared_buff);
		shared_buff = new_buff;
	
	}//if

	keyval_pos_t *kv_p = (keyval_pos_t *)((char *)shared_buff + shared_buff_len - sizeof(keyval_pos_t)*(shared_arr_len + 1));
	kv_p->keySize = keySize;
	kv_p->valSize = valSize;
	kv_p->task_idx = map_task_idx;
	kv_p->next_idx = _COMBINE;				//merged results

	memcpy( (char*)shared_buff + *kv_arr_p->shared_buff_pos, key, keySize);
	kv_p->keyPos = *kv_arr_p->shared_buff_pos;
	*kv_arr_p->shared_buff_pos += (keySize+3)/4*4;

	memcpy( (char*)shared_buff + *kv_arr_p->shared_buff_pos, val, valSize);
	kv_p->valPos = *kv_arr_p->shared_buff_pos;
	*kv_arr_p->shared_buff_pos += (valSize+3)/4*4;
	
	(*kv_arr_p->shared_arr_len)++;

}//void

void* PandaThreadExecuteMapOnCPU(void * ptr)
{

	panda_cpu_task_info_t *panda_cpu_task_info = (panda_cpu_task_info_t *)ptr;
	panda_cpu_context  *pcc = (panda_cpu_context *) (panda_cpu_task_info->pcc);
	//panda_node_context *pnc = (panda_node_context *)(panda_cpu_task_info->pnc);
	
	int start_row_idx	=	panda_cpu_task_info->start_row_idx;
	int end_row_idx		=	panda_cpu_task_info->end_row_idx;

	if(end_row_idx<=start_row_idx) 	return NULL;
	
	char *buff		=	(char *)malloc(sizeof(char)*CPU_SHARED_BUFF_SIZE);
	int *int_arr	=	(int *)malloc(sizeof(int)*(end_row_idx - start_row_idx + 3));
	int *buddy		=	int_arr+3;
	
	int buddy_len	=	end_row_idx	- start_row_idx;
	for (int i=0;i<buddy_len;i++){
		buddy [i]	=	i + start_row_idx;
	}//for
	
	for (int map_idx = start_row_idx; map_idx < end_row_idx; map_idx++){

		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buff)		= buff;
		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buff_len) = int_arr;
		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buff_pos) = int_arr+1;
		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_arr_len)	= int_arr+2;
		
		*(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buff_len)	= CPU_SHARED_BUFF_SIZE;
		*(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buff_pos)	= 0;
		*(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_arr_len)		= 0;
		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buddy)		= buddy;
		(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_idx].shared_buddy_len)	= buddy_len;

	}//for

	for (int map_idx = panda_cpu_task_info->start_row_idx; map_idx < panda_cpu_task_info->end_row_idx; map_idx++){

		keyval_t *kv_p = (keyval_t *)(&(pcc->input_key_vals.input_keyval_arr[map_idx]));
		panda_cpu_map(kv_p->key,kv_p->val,kv_p->keySize,kv_p->valSize,pcc,map_idx);

	}//for
	
	return NULL;
}//int 




void PandaEmitMapOutputOnCPU(void *key, void *val, int keySize, int valSize, panda_cpu_context *pcc, int map_task_idx){
	
	if(map_task_idx >= pcc->input_key_vals.num_input_record) {	ShowError("error ! map_task_idx >= d_g_state->num_input_record");		return;	}
	keyval_arr_t *kv_arr_p = &(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[map_task_idx]);
	
	char *buff = (char*)(kv_arr_p->shared_buff);
	
	if (!((*kv_arr_p->shared_buff_pos) + keySize + valSize < (*kv_arr_p->shared_buff_len) - sizeof(keyval_pos_t)*((*kv_arr_p->shared_arr_len)+1))){
		ShowWarn("Warning! not enough memory at CPU task:%d *kv_arr_p->shared_arr_len:%d current buff_size:%d KB",
			map_task_idx,*kv_arr_p->shared_arr_len,(*kv_arr_p->shared_buff_len)/1024);

		char *new_buff = (char*)malloc(sizeof(char)*((*kv_arr_p->shared_buff_len)*2));
		if(new_buff==NULL){ ShowError("Error ! There is not enough memory to allocat!"); return; }

		memcpy(new_buff, buff, sizeof(char)*(*kv_arr_p->shared_buff_pos));
		int blockSize = sizeof(keyval_pos_t)*(*kv_arr_p->shared_arr_len);
		memcpy(new_buff + (*kv_arr_p->shared_buff_len)*2 - blockSize, 
			(char*)buff + (*kv_arr_p->shared_buff_len) - blockSize,
														blockSize);
		
		(*kv_arr_p->shared_buff_len) = 2*(*kv_arr_p->shared_buff_len);
		for(int  idx = 0; idx < (kv_arr_p->shared_buddy_len); idx++){
			int cur_map_task_idx = kv_arr_p->shared_buddy[idx];  //the buddy relationship won't be changed
			keyval_arr_t *cur_kv_arr_p = &(pcc->intermediate_key_vals.intermediate_keyval_arr_arr_p[cur_map_task_idx]);
			cur_kv_arr_p->shared_buff = new_buff;
		}//for
		free(buff);//
		buff = new_buff;
	}//if
	
	keyval_pos_t *kv_p = (keyval_pos_t *)((char *)buff + *kv_arr_p->shared_buff_len - sizeof(keyval_pos_t)*((*kv_arr_p->shared_arr_len)+1));
	(*kv_arr_p->shared_arr_len)++;
	kv_p->task_idx = map_task_idx;
	kv_p->next_idx = _MAP;
	
	kv_p->keyPos = (*kv_arr_p->shared_buff_pos);
	*kv_arr_p->shared_buff_pos += ((keySize+3)/4)*4;		//alignment 4 bytes for reading and writing
	memcpy((char *)(buff) + kv_p->keyPos, key, keySize);
	kv_p->keySize = keySize;
	
	kv_p->valPos = (*kv_arr_p->shared_buff_pos);
	*kv_arr_p->shared_buff_pos += ((valSize+3)/4)*4;
	//char *val_p = (char *)(buff) + kv_p->valPos;
	memcpy((char *)(buff) + kv_p->valPos, val, valSize);
	kv_p->valSize = valSize;
	(kv_arr_p->arr) = kv_p;
	
}//__device__

#endif //__PANDALIB_CU__
