/*	

Copyright 2012 The Trustees of Indiana University.  All rights reserved.
CGL MapReduce Framework on GPUs and CPUs
Code Name: Panda 0.3
File: main.cu 
Time: 2012-12-10 
Developer: Hui Li (lihui@indiana.edu)

This is the source code for Panda, a MapReduce runtime on GPUs and CPUs.

*/

#include "Panda.h"
#include "UserAPI.h"
#include <ctype.h>


//-----------------------------------------------------------------------
//usage: C-means datafile
//param: datafile 
//-----------------------------------------------------------------------


static float *GenPointsFloat(int numPt, int dim)
{
	float *matrix = (float*)malloc(sizeof(float)*numPt*dim);
	srand(time(0));
	for (int i = 0; i < numPt; i++)
		for (int j = 0; j < dim; j++)
			matrix[i*dim+j] = (float)((rand() % 100)/73.0);
	return matrix;
}//static float 

static float *GenInitCentersFloat(float* points, int numPt, int dim, int K)
{
	float* centers = (float*)malloc(sizeof(float)*K*dim);

	for (int i = 0; i < K; i++)
		for (int j = 0; j < dim; j++)
			centers[i*dim+j] = points[i*dim + j];
	return centers;
}//

int main(int argc, char** argv) 
{		
	if (argc != 7)
	{
		printf("Panda C-means\n");
		printf("usage: %s [numPt]  [cpu/gpu ratio] [numMapperPerCPU][numMapperPerGPU] [numGPU] [numCPU]\n", argv[0]);
		exit(-1);//[Dimensions] [numClusters]
	}//if
	
	//printf("start %s  %s  %s\n",argv[0],argv[1],argv[2]);
	int numPt = atoi(argv[1]);
	int dim = 100;//atoi(argv[2]);
	int K = 10;//atoi(argv[3]);
	float ratio = atof(argv[2]);
	int numMapperCPU = atoi(argv[3]);
	int numMapperGPU = atoi(argv[4]);
	
	int maxIter = 1;//atoi(argv[5]);

	numMapperGPU = 1;
	numMapperCPU = 1;

	int num_gpu_core_groups = 0;
	int num_gpu_card_groups = 1;
	int num_cpus_groups = 0;

	panda_context *panda = CreatePandaContext();
	panda->num_gpu_core_groups = num_gpu_core_groups;
	panda->num_gpu_card_groups = num_gpu_card_groups;
	panda->num_cpus_groups = num_cpus_groups;

	panda->cpu_ratio = 0.0;
	panda->gpu_card_ratio = 1.0;
	panda->gpu_core_ratio = 0.0;

	ShowLog("numPt:%d	dim:%d	K:%d	numMapperGPU:%d numMapperCPU:%d	maxIter:%d",numPt,dim,K,numMapperGPU,numMapperCPU,maxIter);
	float* h_points = GenPointsFloat(numPt, dim);
	float* h_cluster = GenInitCentersFloat(h_points, numPt, dim, K);
	
	int numgpus = 0;
	hipGetDeviceCount(&numgpus);
	if (num_gpu_core_groups > numgpus)		num_gpu_core_groups = numgpus;
	if (num_gpu_card_groups > numgpus-num_gpu_core_groups) num_gpu_card_groups = numgpus-num_gpu_core_groups;
			
	pthread_t *no_threads = (pthread_t*)malloc(sizeof(pthread_t)*(num_gpu_core_groups+num_gpu_card_groups+num_cpus_groups));
	thread_info_t *thread_info = (thread_info_t*)malloc(sizeof(thread_info_t)*(num_gpu_core_groups+num_gpu_card_groups+num_cpus_groups));
		
	int gpuCoreWorkLoad = int(numPt * panda->gpu_core_ratio);
	int gpuCardWorkLoad = int(numPt * panda->gpu_card_ratio);
	int cpuWorkLoad = int(numPt * panda->cpu_ratio);

	gpuCardWorkLoad = numPt;
	gpuCoreWorkLoad = 0;
	cpuWorkLoad = 0;

	/*if (num_gpu_card_groups == 0) {
		gpuCardWorkLoad = 0;
		ratio = 1.0;
	}
	if (num_gpu_core_groups == 0) {
		gpuCoreWorkLoad = 0;
		ratio = 1.0;
	}
	if (num_cpus_groups == 0){
		cpuWorkLoad = 0;
		ratio = 0.0;
	}*/


	ShowLog("numPt:%d cpuWorkload:%d gpuCoreWorkload:%d gpuCardWorkload:%d", numPt, cpuWorkLoad, gpuCoreWorkLoad, gpuCardWorkLoad);
	int global_dev_id = 0;

	for (int dev_id=0; dev_id<num_gpu_core_groups; dev_id++, global_dev_id++){
		
		job_configuration *gpu_job_conf = CreateJobConf();
		gpu_job_conf->num_gpu_core_groups = num_gpu_core_groups;
		gpu_job_conf->num_mappers = numMapperGPU;
		gpu_job_conf->auto_tuning = false;
		//gpu_job_conf->cpu_ratio = (double)ratio;
		gpu_job_conf->auto_tuning_sample_rate = -1;//sample_rate;
		gpu_job_conf->iterative_support = false;
		//gpu_job_conf->local_combiner = true;

		int tid = global_dev_id;		
		float* d_points	=	NULL;
		float* d_cluster =	NULL;
		//int* d_change	=	NULL;
		int* d_clusterId =	NULL;
		
		float* d_tempClusters = NULL;
		float* d_tempDenominators = NULL;
		
		if (tid<numgpus)
		checkCudaErrors(hipSetDevice(tid));

		checkCudaErrors(hipMalloc((void**)&d_points, numPt*dim*sizeof(int)));
		checkCudaErrors(hipMemcpy(d_points, h_points, numPt*dim*sizeof(int), hipMemcpyHostToDevice));
		checkCudaErrors(hipMalloc((void**)&d_clusterId, numPt*sizeof(int)));
		checkCudaErrors(hipMemset(d_clusterId, 0, numPt*sizeof(int)));
		checkCudaErrors(hipMalloc((void**)&d_cluster, K*dim*sizeof(int)));
		checkCudaErrors(hipMemcpy(d_cluster, h_cluster, K*dim*sizeof(int), hipMemcpyHostToDevice));
		//checkCudaErrors(hipMalloc((void**)&d_change, sizeof(int)));
		//checkCudaErrors(hipMemset(d_change, 0, sizeof(int)));
		checkCudaErrors(hipMalloc((void**)&d_tempClusters,K*dim*numMapperGPU*sizeof(float)));
		checkCudaErrors(hipMemset(d_tempClusters, 0, sizeof(float)*K*dim*numMapperGPU));
		checkCudaErrors(hipMalloc((void**)&d_tempDenominators,numMapperGPU * K * sizeof(float)));
		
		checkCudaErrors(hipMemset(d_tempDenominators, 0, sizeof(float)*K*numMapperGPU));
		thread_info[global_dev_id].tid = global_dev_id;
		//thread_info[dev_id].num_gpu_core_groups = num_gpu_core_groups;
		thread_info[global_dev_id].device_type = GPU_CORE_ACC;
		
		hipDeviceProp_t gpu_dev;
		hipGetDeviceProperties(&gpu_dev, dev_id);
		ShowLog("Configure Device ID:%d: Device Name:%s", dev_id, gpu_dev.name);
		thread_info[dev_id].device_name = gpu_dev.name;
				
		KM_VAL_T val;
		//val.ptrPoints = (int *)d_points;
		//val.ptrClusters = (int *)d_cluster;
		val.d_Points = d_points;
		val.d_Clusters = d_cluster;
		//val.ptrChange = d_change;
		
		KM_KEY_T key;
		key.dim = dim;
		key.K = K;
		//key.ptrClusterId = d_clusterId;
		
		int numPtPerGPU = gpuCoreWorkLoad/num_gpu_core_groups;
		int start = dev_id*numPtPerGPU;
		int end = start+numPtPerGPU;
		if (dev_id==num_gpu_core_groups-1)
			end = gpuCoreWorkLoad;
		
		int numPtPerMap = (end-start)/numMapperGPU;
		ShowLog("GPU core numPtPerMap:%d startPt:%d  endPt:%d numPt:%d gpuCoreWorkLoad:%d",numPtPerMap,start,end,numPt,gpuCoreWorkLoad);

		int start_i,end_i;
		start_i = start;
		for (int j = 0; j < numMapperGPU; j++)
		{	
			end_i = start_i + numPtPerMap;
			if ( j < (end-start)%numMapperGPU)
				end_i++;
			
			//ShowLog("start_i:%d, start_j:%d",start_i,end_i);
			//key.point_id = start_i;
			key.start = start_i;
			key.end = end_i;
			key.global_map_id = dev_id*numMapperGPU+j;
			key.local_map_id = j;

			val.d_Points = d_points;
			val.d_tempDenominators = d_tempDenominators;
			val.d_tempClusters = d_tempClusters;

			AddPandaTask(gpu_job_conf, &key, &val, sizeof(KM_KEY_T), sizeof(KM_VAL_T));
			start_i = end_i;
		}//for
		thread_info[dev_id].job_conf = gpu_job_conf;
		thread_info[dev_id].device_type = GPU_CORE_ACC;
	}//for

	for (int dev_id = 0; dev_id < num_gpu_card_groups; dev_id++, global_dev_id++){

		job_configuration *gpu_job_conf = CreateJobConf();
		gpu_job_conf->num_gpu_card_groups = num_gpu_card_groups;
		gpu_job_conf->num_mappers = numgpus;

		gpu_job_conf->auto_tuning = false;
		//gpu_job_conf->cpu_ratio = (double)ratio;
		gpu_job_conf->auto_tuning_sample_rate = -1;//sample_rate;
		gpu_job_conf->iterative_support = false;
		//gpu_job_conf->local_combiner = true;

		int tid = global_dev_id;	
		KM_VAL_T val;
		//val.ptrPoints = (int *)d_points;
		//val.ptrClusters = (int *)d_cluster;
		//val.d_Points = d_points;
		//val.d_Clusters = d_cluster;
		//val.d_distanceMatrix = d_distanceMatrix;

		//TODO
		val.d_Points = h_points;
		val.d_Clusters = h_cluster;

		//val.ptrChange = d_change;
		
		KM_KEY_T key;
		key.dim = dim;
		key.K = K;
		//key.ptrClusterId = d_clusterId;
		
		int numPtPerGPU = gpuCoreWorkLoad/num_gpu_card_groups;
		int start = dev_id*numPtPerGPU;
		int end = start+numPtPerGPU;
		if (dev_id==num_gpu_card_groups-1)
			end = gpuCardWorkLoad;
		
		int numPtPerMap = (end-start);
		ShowLog("GPU card numPtPerMap:%d startPt:%d  endPt:%d numPt:%d gpuCardWorkLoad:%d",numPtPerMap,start,end,numPt,gpuCardWorkLoad);

		float* h_tempClusters = NULL;
		float* h_tempDenominators = NULL;
		h_tempClusters = (float *)malloc(K*dim*numMapperGPU*sizeof(float));
		h_tempDenominators = (float *)malloc(numMapperGPU*K*sizeof(float));

		int start_i,end_i;
		start_i = start;

		//one map task for each gpu card group
		for (int j = 0; j < 1; j++)
		{	
			end_i = start_i + numPtPerMap;
			//ShowLog("start_i:%d, start_j:%d",start_i,end_i);
			//key.point_id = start_i;
			key.start = start_i;
			key.end = end_i;
			key.global_map_id = global_dev_id;
			key.local_map_id = j;

			//val.d_Points = h_points;
			val.d_tempDenominators = h_tempDenominators;
			val.d_tempClusters = h_tempClusters;

			

			AddPandaTask(gpu_job_conf, &key, &val, sizeof(KM_KEY_T), sizeof(KM_VAL_T));
			start_i = end_i;
		}//for
		thread_info[dev_id].job_conf = gpu_job_conf;
		thread_info[dev_id].device_type = GPU_CARD_ACC;

	}//for

	for (int dev_id = 0; dev_id < num_cpus_groups; dev_id++, global_dev_id++){

		job_configuration *cpu_job_conf = CreateJobConf();
		cpu_job_conf->num_cpus_groups = num_cpus_groups;
		cpu_job_conf->num_cpus_cores = getCPUCoresNum();
		//cpu_job_conf->local_combiner = true;
		//cpu_job_conf->cpu_ratio = (double)ratio;
		cpu_job_conf->auto_tuning_sample_rate = -1;
		cpu_job_conf->iterative_support = false;
		cpu_job_conf->local_combiner = false;

		int tid = dev_id;		

		
		float* h_tempClusters = NULL;
		float* h_tempDenominators = NULL;

		h_tempClusters = (float *)malloc(K*dim*numMapperGPU*sizeof(float));
		h_tempDenominators = (float *)malloc(numMapperGPU*K*sizeof(float));
		
		thread_info[dev_id].device_name = "CPU";
			
		KM_VAL_T val;
		
		val.d_Points = h_points;
		val.d_Clusters = h_cluster;
		//val.ptrChange = d_change;
		
		KM_KEY_T key;
		key.dim = dim;
		key.K = K;
		//int * h_clusterId = (int*)malloc(sizeof(int)*numPt);
		//key.ptrClusterId = h_clusterId;
		
		int numPtPerCPU = cpuWorkLoad/num_cpus_groups;
		int start = gpuCoreWorkLoad+ (dev_id-num_gpu_core_groups)*numPtPerCPU;
		int end = start+numPtPerCPU;

		if (dev_id==num_gpu_core_groups+num_cpus_groups-1)
			end = numPt;
		
		int numPtPerMap = (end-start)/numMapperCPU;
		ShowLog("CPU numPtPerMap:%d startPt:%d  endPt:%d numPt:%d cpuWorkLoad:%d",numPtPerMap,start,end,numPt,cpuWorkLoad);

		int start_i, end_i;
		start_i = start;

		if (cpuWorkLoad>0)
		for (int j = 0; j < numMapperCPU; j++)
		{	
			end_i = start_i + numPtPerMap;
			if ( j < (end-start)%numMapperCPU )
				end_i++;

			if(end_i==start_i)		break;

			//ShowLog("start_i:%d, start_j:%d",start_i,end_i);
			//key.point_id = start_i;
			key.start = start_i;
			key.end = end_i;
			key.global_map_id = dev_id*numMapperCPU+j;
			key.local_map_id = j;

			//val.d_Points = h_points;
			val.d_tempDenominators = h_tempDenominators;
			val.d_tempClusters = h_tempClusters;

			AddPandaTask(cpu_job_conf, &key, &val, sizeof(KM_KEY_T), sizeof(KM_VAL_T));
			start_i = end_i;
		}//for
		
		thread_info[dev_id].job_conf = cpu_job_conf;
		thread_info[dev_id].device_type = CPU_ACC;
		thread_info[dev_id].tid = dev_id;
		
	}//for


	double t1 = PandaTimer();
	int iter = 0;
	while (iter<maxIter)
	{
/////////////////////////////////////////////////////////
		PandaMetaScheduler(thread_info, panda);
/////////////////////////////////////////////////////////
		hipDeviceSynchronize();
		iter++;
	}//while iterations
	double t2 = PandaTimer();
	ShowLog("Panda C-means take %f sec", t2-t1);
	DoLog2Disk("== Panda C-means numPt:%d	dim:%d	K:%d	numMapperGPU:%d	numMapperCPU:%d maxIter:%d take %f sec",numPt,dim,K,numMapperGPU,numMapperCPU,maxIter, t2-t1);

	return 0;
}//		
